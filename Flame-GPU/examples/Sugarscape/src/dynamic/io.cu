
/*
 * FLAME GPU v 1.5.X for CUDA 9
 * Copyright University of Sheffield.
 * Original Author: Dr Paul Richmond (user contributions tracked on https://github.com/FLAMEGPU/FLAMEGPU)
 * Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
 *
 * University of Sheffield retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * University of Sheffield is strictly prohibited.
 *
 * For terms of licence agreement please attached licence or view licence
 * on www.flamegpu.com website.
 *
 */


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cmath>
#include <limits.h>
#include <algorithm>
#include <string>
#include <vector>



#ifdef _WIN32
#define strtok_r strtok_s
#endif

// include header
#include "header.h"

glm::vec3 agent_maximum;
glm::vec3 agent_minimum;

int fpgu_strtol(const char* str){
    return (int)strtol(str, NULL, 0);
}

unsigned int fpgu_strtoul(const char* str){
    return (unsigned int)strtoul(str, NULL, 0);
}

long long int fpgu_strtoll(const char* str){
    return strtoll(str, NULL, 0);
}

unsigned long long int fpgu_strtoull(const char* str){
    return strtoull(str, NULL, 0);
}

double fpgu_strtod(const char* str){
    return strtod(str, NULL);
}

float fgpu_atof(const char* str){
    return (float)atof(str);
}


//templated class function to read array inputs from supported types
template <class T>
void readArrayInput( T (*parseFunc)(const char*), char* buffer, T *array, unsigned int expected_items){
    unsigned int i = 0;
    const char s[2] = ",";
    char * token;
    char * end_str;

    token = strtok_r(buffer, s, &end_str);
    while (token != NULL){
        if (i>=expected_items){
            printf("Error: variable array has too many items, expected %d!\n", expected_items);
            exit(EXIT_FAILURE);
        }
        
        array[i++] = (T)parseFunc(token);
        
        token = strtok_r(NULL, s, &end_str);
    }
    if (i != expected_items){
        printf("Error: variable array has %d items, expected %d!\n", i, expected_items);
        exit(EXIT_FAILURE);
    }
}

//templated class function to read array inputs from supported types
template <class T, class BASE_T, unsigned int D>
void readArrayInputVectorType( BASE_T (*parseFunc)(const char*), char* buffer, T *array, unsigned int expected_items){
    unsigned int i = 0;
    const char s[2] = "|";
    char * token;
    char * end_str;

    token = strtok_r(buffer, s, &end_str);
    while (token != NULL){
        if (i>=expected_items){
            printf("Error: Agent memory array has too many items, expected %d!\n", expected_items);
            exit(EXIT_FAILURE);
        }
        
        //read vector type as an array
        T vec;
        readArrayInput<BASE_T>(parseFunc, token, (BASE_T*) &vec, D);
        array[i++] = vec;
        
        token = strtok_r(NULL, s, &end_str);
    }
    if (i != expected_items){
        printf("Error: Agent memory array has %d items, expected %d!\n", i, expected_items);
        exit(EXIT_FAILURE);
    }
}

void saveIterationData(char* outputpath, int iteration_number, xmachine_memory_agent_list* h_agents_default, xmachine_memory_agent_list* d_agents_default, int h_xmachine_memory_agent_default_count)
{
    PROFILE_SCOPED_RANGE("saveIterationData");
	hipError_t cudaStatus;
	
	//Device to host memory transfer
	
	cudaStatus = hipMemcpy( h_agents_default, d_agents_default, sizeof(xmachine_memory_agent_list), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr,"Error Copying agent Agent default State Memory from GPU: %s\n", hipGetErrorString(cudaStatus));
		exit(cudaStatus);
	}
	
	/* Pointer to file */
	FILE *file;
	char data[100];

	sprintf(data, "%s%i.xml", outputpath, iteration_number);
	//printf("Writing iteration %i data to %s\n", iteration_number, data);
	file = fopen(data, "w");
    if(file == nullptr){
        printf("Error: Could not open file `%s` for output. Aborting.\n", data);
        exit(EXIT_FAILURE);
    }
    fputs("<states>\n<itno>", file);
    sprintf(data, "%i", iteration_number);
    fputs(data, file);
    fputs("</itno>\n", file);
    fputs("<environment>\n" , file);
    
	fputs("</environment>\n" , file);

	//Write each agent agent to xml
	for (int i=0; i<h_xmachine_memory_agent_default_count; i++){
		fputs("<xagent>\n" , file);
		fputs("<name>agent</name>\n", file);
        
		fputs("<location_id>", file);
        sprintf(data, "%d", h_agents_default->location_id[i]);
		fputs(data, file);
		fputs("</location_id>\n", file);
        
		fputs("<agent_id>", file);
        sprintf(data, "%d", h_agents_default->agent_id[i]);
		fputs(data, file);
		fputs("</agent_id>\n", file);
        
		fputs("<state>", file);
        sprintf(data, "%d", h_agents_default->state[i]);
		fputs(data, file);
		fputs("</state>\n", file);
        
		fputs("<sugar_level>", file);
        sprintf(data, "%d", h_agents_default->sugar_level[i]);
		fputs(data, file);
		fputs("</sugar_level>\n", file);
        
		fputs("<metabolism>", file);
        sprintf(data, "%d", h_agents_default->metabolism[i]);
		fputs(data, file);
		fputs("</metabolism>\n", file);
        
		fputs("<env_sugar_level>", file);
        sprintf(data, "%d", h_agents_default->env_sugar_level[i]);
		fputs(data, file);
		fputs("</env_sugar_level>\n", file);
        
		fputs("</xagent>\n", file);
	}
	
	

	fputs("</states>\n" , file);
	
	/* Close the file */
	fclose(file);

}

void readInitialStates(char* inputpath, xmachine_memory_agent_list* h_agents, int* h_xmachine_memory_agent_count)
{
    PROFILE_SCOPED_RANGE("readInitialStates");

	int temp = 0;
	int* itno = &temp;

	/* Pointer to file */
	FILE *file;
	/* Char and char buffer for reading file to */
	char c = ' ';
	const int bufferSize = 10000;
	char buffer[bufferSize];
	char agentname[1000];

	/* Pointer to x-memory for initial state data */
	/*xmachine * current_xmachine;*/
	/* Variables for checking tags */
	int reading, i;
	int in_tag, in_itno, in_xagent, in_name, in_comment;
    int in_agent_location_id;
    int in_agent_agent_id;
    int in_agent_state;
    int in_agent_sugar_level;
    int in_agent_metabolism;
    int in_agent_env_sugar_level;
    
    /* tags for environment global variables */
    int in_env;
	/* set agent count to zero */
	*h_xmachine_memory_agent_count = 0;
	
	/* Variables for initial state data */
	int agent_location_id;
	int agent_agent_id;
	int agent_state;
	int agent_sugar_level;
	int agent_metabolism;
	int agent_env_sugar_level;

    /* Variables for environment variables */
    


	/* Initialise variables */
    agent_maximum.x = 0;
    agent_maximum.y = 0;
    agent_maximum.z = 0;
    agent_minimum.x = 0;
    agent_minimum.y = 0;
    agent_minimum.z = 0;
	reading = 1;
    in_comment = 0;
	in_tag = 0;
	in_itno = 0;
    in_env = 0;
    in_xagent = 0;
	in_name = 0;
	in_agent_location_id = 0;
	in_agent_agent_id = 0;
	in_agent_state = 0;
	in_agent_sugar_level = 0;
	in_agent_metabolism = 0;
	in_agent_env_sugar_level = 0;
	//set all agent values to 0
	//If this is not done then it will cause errors in emu mode where undefined memory is not 0
	for (int k=0; k<xmachine_memory_agent_MAX; k++)
	{	
		h_agents->location_id[k] = 0;
		h_agents->agent_id[k] = 0;
		h_agents->state[k] = 0;
		h_agents->sugar_level[k] = 0;
		h_agents->metabolism[k] = 0;
		h_agents->env_sugar_level[k] = 0;
	}
	

	/* Default variables for memory */
    agent_location_id = 0;
    agent_agent_id = 0;
    agent_state = 0;
    agent_sugar_level = 0;
    agent_metabolism = 0;
    agent_env_sugar_level = 0;

    /* Default variables for environment variables */
    
    
    // If no input path was specified, issue a message and return.
    if(inputpath[0] == '\0'){
        printf("No initial states file specified. Using default values.\n");
        return;
    }
    
    // Otherwise an input path was specified, and we have previously checked that it is (was) not a directory. 
    
	// Attempt to open the non directory path as read only.
	file = fopen(inputpath, "r");
    
    // If the file could not be opened, issue a message and return.
    if(file == nullptr)
    {
      printf("Could not open input file %s. Continuing with default values\n", inputpath);
      return;
    }
    // Otherwise we can iterate the file until the end of XML is reached.
    size_t bytesRead = 0;
    i = 0;
	while(reading==1)
	{
        // If I exceeds our buffer size we must abort
        if(i >= bufferSize){
            fprintf(stderr, "Error: XML Parsing failed Tag name or content too long (> %d characters)\n", bufferSize);
            exit(EXIT_FAILURE);
        }

		/* Get the next char from the file */
		c = (char)fgetc(file);

        // Check if we reached the end of the file.
        if(c == EOF){
            // Break out of the loop. This allows for empty files(which may or may not be)
            break;
        }
        // Increment byte counter.
        bytesRead++;

        /*If in a  comment, look for the end of a comment */
        if(in_comment){

            /* Look for an end tag following two (or more) hyphens.
               To support very long comments, we use the minimal amount of buffer we can. 
               If we see a hyphen, store it and increment i (but don't increment i)
               If we see a > check if we have a correct terminating comment
               If we see any other characters, reset i.
            */

            if(c == '-'){
                buffer[i] = c;
                i++;
            } else if(c == '>' && i >= 2){
                in_comment = 0;
                i = 0;
            } else {
                i = 0;
            }

            /*// If we see the end tag, check the preceding two characters for a close comment, if enough characters have been read for -->
            if(c == '>' && i >= 2 && buffer[i-1] == '-' && buffer[i-2] == '-'){
                in_comment = 0;
                buffer[0] = 0;
                i = 0;
            } else {
                // Otherwise just store it in the buffer so we can keep checking for close tags
                buffer[i] = c;
                i++;
            }*/
        }
		/* If the end of a tag */
		else if(c == '>')
		{
			/* Place 0 at end of buffer to make chars a string */
			buffer[i] = 0;

			if(strcmp(buffer, "states") == 0) reading = 1;
			if(strcmp(buffer, "/states") == 0) reading = 0;
			if(strcmp(buffer, "itno") == 0) in_itno = 1;
			if(strcmp(buffer, "/itno") == 0) in_itno = 0;
            if(strcmp(buffer, "environment") == 0) in_env = 1;
            if(strcmp(buffer, "/environment") == 0) in_env = 0;
			if(strcmp(buffer, "name") == 0) in_name = 1;
			if(strcmp(buffer, "/name") == 0) in_name = 0;
            if(strcmp(buffer, "xagent") == 0) in_xagent = 1;
			if(strcmp(buffer, "/xagent") == 0)
			{
				if(strcmp(agentname, "agent") == 0)
				{
					if (*h_xmachine_memory_agent_count > xmachine_memory_agent_MAX){
						printf("ERROR: MAX Buffer size (%i) for agent agent exceeded whilst reading data\n", xmachine_memory_agent_MAX);
						// Close the file and stop reading
						fclose(file);
						exit(EXIT_FAILURE);
					}
                    
					h_agents->location_id[*h_xmachine_memory_agent_count] = agent_location_id;
					h_agents->agent_id[*h_xmachine_memory_agent_count] = agent_agent_id;
					h_agents->state[*h_xmachine_memory_agent_count] = agent_state;
					h_agents->sugar_level[*h_xmachine_memory_agent_count] = agent_sugar_level;
					h_agents->metabolism[*h_xmachine_memory_agent_count] = agent_metabolism;
					h_agents->env_sugar_level[*h_xmachine_memory_agent_count] = agent_env_sugar_level;
					(*h_xmachine_memory_agent_count) ++;	
				}
				else
				{
					printf("Warning: agent name undefined - '%s'\n", agentname);
				}



				/* Reset xagent variables */
                agent_location_id = 0;
                agent_agent_id = 0;
                agent_state = 0;
                agent_sugar_level = 0;
                agent_metabolism = 0;
                agent_env_sugar_level = 0;
                
                in_xagent = 0;
			}
			if(strcmp(buffer, "location_id") == 0) in_agent_location_id = 1;
			if(strcmp(buffer, "/location_id") == 0) in_agent_location_id = 0;
			if(strcmp(buffer, "agent_id") == 0) in_agent_agent_id = 1;
			if(strcmp(buffer, "/agent_id") == 0) in_agent_agent_id = 0;
			if(strcmp(buffer, "state") == 0) in_agent_state = 1;
			if(strcmp(buffer, "/state") == 0) in_agent_state = 0;
			if(strcmp(buffer, "sugar_level") == 0) in_agent_sugar_level = 1;
			if(strcmp(buffer, "/sugar_level") == 0) in_agent_sugar_level = 0;
			if(strcmp(buffer, "metabolism") == 0) in_agent_metabolism = 1;
			if(strcmp(buffer, "/metabolism") == 0) in_agent_metabolism = 0;
			if(strcmp(buffer, "env_sugar_level") == 0) in_agent_env_sugar_level = 1;
			if(strcmp(buffer, "/env_sugar_level") == 0) in_agent_env_sugar_level = 0;
			
            /* environment variables */
            

			/* End of tag and reset buffer */
			in_tag = 0;
			i = 0;
		}
		/* If start of tag */
		else if(c == '<')
		{
			/* Place /0 at end of buffer to end numbers */
			buffer[i] = 0;
			/* Flag in tag */
			in_tag = 1;

			if(in_itno) *itno = atoi(buffer);
			if(in_name) strcpy(agentname, buffer);
			else if (in_xagent)
			{
				if(in_agent_location_id){
                    agent_location_id = (int) fpgu_strtol(buffer); 
                }
				if(in_agent_agent_id){
                    agent_agent_id = (int) fpgu_strtol(buffer); 
                }
				if(in_agent_state){
                    agent_state = (int) fpgu_strtol(buffer); 
                }
				if(in_agent_sugar_level){
                    agent_sugar_level = (int) fpgu_strtol(buffer); 
                }
				if(in_agent_metabolism){
                    agent_metabolism = (int) fpgu_strtol(buffer); 
                }
				if(in_agent_env_sugar_level){
                    agent_env_sugar_level = (int) fpgu_strtol(buffer); 
                }
				
            }
            else if (in_env){
            
            }
		/* Reset buffer */
			i = 0;
		}
		/* If in tag put read char into buffer */
		else if(in_tag)
		{
            // Check if we are a comment, when we are in a tag and buffer[0:2] == "!--"
            if(i == 2 && c == '-' && buffer[1] == '-' && buffer[0] == '!'){
                in_comment = 1;
                // Reset the buffer and i.
                buffer[0] = 0;
                i = 0;
            }

            // Store the character and increment the counter
            buffer[i] = c;
            i++;

		}
		/* If in data read char into buffer */
		else
		{
			buffer[i] = c;
			i++;
		}
	}
    // If no bytes were read, raise a warning.
    if(bytesRead == 0){
        fprintf(stdout, "Warning: %s is an empty file\n", inputpath);
        fflush(stdout);
    }

    // If the in_comment flag is still marked, issue a warning.
    if(in_comment){
        fprintf(stdout, "Warning: Un-terminated comment in %s\n", inputpath);
        fflush(stdout);
    }    

	/* Close the file */
	fclose(file);
}

glm::vec3 getMaximumBounds(){
    return agent_maximum;
}

glm::vec3 getMinimumBounds(){
    return agent_minimum;
}


/* Methods to load static networks from disk */
