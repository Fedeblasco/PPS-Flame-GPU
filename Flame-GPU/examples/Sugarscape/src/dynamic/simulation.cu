#include "hip/hip_runtime.h"

/*
 * FLAME GPU v 1.5.X for CUDA 9
 * Copyright University of Sheffield.
 * Original Author: Dr Paul Richmond (user contributions tracked on https://github.com/FLAMEGPU/FLAMEGPU)
 * Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
 *
 * University of Sheffield retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * University of Sheffield is strictly prohibited.
 *
 * For terms of licence agreement please attached licence or view licence
 * on www.flamegpu.com website.
 *
 */


  //Disable internal thrust warnings about conversions
  #ifdef _MSC_VER
  #pragma warning(push)
  #pragma warning (disable : 4267)
  #pragma warning (disable : 4244)
  #endif
  #ifdef __GNUC__
  #pragma GCC diagnostic push
  #pragma GCC diagnostic ignored "-Wunused-parameter"
  #endif

  // includes
  #include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cmath>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/extrema.h>
#include <thrust/system/cuda/execution_policy.h>
#include <hipcub/hipcub.hpp>

// include FLAME kernels
#include "FLAMEGPU_kernals.cu"


#ifdef _MSC_VER
#pragma warning(pop)
#endif
#ifdef __GNUC__
#pragma GCC diagnostic pop
#endif

/* Error check function for safe CUDA API calling */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* Error check function for post CUDA Kernel calling */
#define gpuErrchkLaunch() { gpuLaunchAssert(__FILE__, __LINE__); }
inline void gpuLaunchAssert(const char *file, int line, bool abort=true)
{
	gpuAssert( hipPeekAtLastError(), file, line );
#ifdef _DEBUG
	gpuAssert( hipDeviceSynchronize(), file, line );
#endif
   
}

/* SM padding and offset variables */
int SM_START;
int PADDING;

unsigned int g_iterationNumber;

/* Agent Memory */

/* agent Agent variables these lists are used in the agent function where as the other lists are used only outside the agent functions*/
xmachine_memory_agent_list* d_agents;      /**< Pointer to agent list (population) on the device*/
xmachine_memory_agent_list* d_agents_swap; /**< Pointer to agent list swap on the device (used when killing agents)*/
xmachine_memory_agent_list* d_agents_new;  /**< Pointer to new agent list on the device (used to hold new agents before they are appended to the population)*/
int h_xmachine_memory_agent_count;   /**< Agent population size counter */ 
int h_xmachine_memory_agent_pop_width;   /**< Agent population width */
uint * d_xmachine_memory_agent_keys;	  /**< Agent sort identifiers keys*/
uint * d_xmachine_memory_agent_values;  /**< Agent sort identifiers value */

/* agent state variables */
xmachine_memory_agent_list* h_agents_default;      /**< Pointer to agent list (population) on host*/
xmachine_memory_agent_list* d_agents_default;      /**< Pointer to agent list (population) on the device*/
int h_xmachine_memory_agent_default_count;   /**< Agent population size counter */ 


/* Variables to track the state of host copies of state lists, for the purposes of host agent data access.
 * @future - if the host data is current it may be possible to avoid duplicating memcpy in xml output.
 */
unsigned int h_agents_default_variable_location_id_data_iteration;
unsigned int h_agents_default_variable_agent_id_data_iteration;
unsigned int h_agents_default_variable_state_data_iteration;
unsigned int h_agents_default_variable_sugar_level_data_iteration;
unsigned int h_agents_default_variable_metabolism_data_iteration;
unsigned int h_agents_default_variable_env_sugar_level_data_iteration;


/* Message Memory */

/* cell_state Message variables */
xmachine_message_cell_state_list* h_cell_states;         /**< Pointer to message list on host*/
xmachine_message_cell_state_list* d_cell_states;         /**< Pointer to message list on device*/
xmachine_message_cell_state_list* d_cell_states_swap;    /**< Pointer to message swap list on device (used for holding optional messages)*/
/* Discrete Partitioning Variables*/
int h_message_cell_state_range;     /**< range of the discrete message*/
int h_message_cell_state_width;     /**< with of the message grid*/
/* Texture offset values for host */
int h_tex_xmachine_message_cell_state_location_id_offset;
int h_tex_xmachine_message_cell_state_state_offset;
int h_tex_xmachine_message_cell_state_env_sugar_level_offset;
/* movement_request Message variables */
xmachine_message_movement_request_list* h_movement_requests;         /**< Pointer to message list on host*/
xmachine_message_movement_request_list* d_movement_requests;         /**< Pointer to message list on device*/
xmachine_message_movement_request_list* d_movement_requests_swap;    /**< Pointer to message swap list on device (used for holding optional messages)*/
/* Discrete Partitioning Variables*/
int h_message_movement_request_range;     /**< range of the discrete message*/
int h_message_movement_request_width;     /**< with of the message grid*/
/* Texture offset values for host */
int h_tex_xmachine_message_movement_request_agent_id_offset;
int h_tex_xmachine_message_movement_request_location_id_offset;
int h_tex_xmachine_message_movement_request_sugar_level_offset;
int h_tex_xmachine_message_movement_request_metabolism_offset;
/* movement_response Message variables */
xmachine_message_movement_response_list* h_movement_responses;         /**< Pointer to message list on host*/
xmachine_message_movement_response_list* d_movement_responses;         /**< Pointer to message list on device*/
xmachine_message_movement_response_list* d_movement_responses_swap;    /**< Pointer to message swap list on device (used for holding optional messages)*/
/* Discrete Partitioning Variables*/
int h_message_movement_response_range;     /**< range of the discrete message*/
int h_message_movement_response_width;     /**< with of the message grid*/
/* Texture offset values for host */
int h_tex_xmachine_message_movement_response_location_id_offset;
int h_tex_xmachine_message_movement_response_agent_id_offset;
  
/* CUDA Streams for function layers */
hipStream_t stream1;

/* Device memory and sizes for CUB values */

void * d_temp_scan_storage_agent;
size_t temp_scan_storage_bytes_agent;


/*Global condition counts*/
int h_metabolise_and_growback_condition_count;


/* RNG rand48 */
RNG_rand48* h_rand48;    /**< Pointer to RNG_rand48 seed list on host*/
RNG_rand48* d_rand48;    /**< Pointer to RNG_rand48 seed list on device*/

/* Cuda Event Timers for Instrumentation */
#if defined(INSTRUMENT_ITERATIONS) && INSTRUMENT_ITERATIONS
	hipEvent_t instrument_iteration_start, instrument_iteration_stop;
	float instrument_iteration_milliseconds = 0.0f;
#endif
#if (defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS) || (defined(INSTRUMENT_INIT_FUNCTIONS) && INSTRUMENT_INIT_FUNCTIONS) || (defined(INSTRUMENT_STEP_FUNCTIONS) && INSTRUMENT_STEP_FUNCTIONS) || (defined(INSTRUMENT_EXIT_FUNCTIONS) && INSTRUMENT_EXIT_FUNCTIONS)
	hipEvent_t instrument_start, instrument_stop;
	float instrument_milliseconds = 0.0f;
#endif

/* CUDA Parallel Primatives variables */
int scan_last_sum;           /**< Indicates if the position (in message list) of last message*/
int scan_last_included;      /**< Indicates if last sum value is included in the total sum count*/

/* Agent function prototypes */

/** agent_metabolise_and_growback
 * Agent function prototype for metabolise_and_growback function of agent agent
 */
void agent_metabolise_and_growback(hipStream_t &stream);

/** agent_output_cell_state
 * Agent function prototype for output_cell_state function of agent agent
 */
void agent_output_cell_state(hipStream_t &stream);

/** agent_movement_request
 * Agent function prototype for movement_request function of agent agent
 */
void agent_movement_request(hipStream_t &stream);

/** agent_movement_response
 * Agent function prototype for movement_response function of agent agent
 */
void agent_movement_response(hipStream_t &stream);

/** agent_movement_transaction
 * Agent function prototype for movement_transaction function of agent agent
 */
void agent_movement_transaction(hipStream_t &stream);

  
void setPaddingAndOffset()
{
    PROFILE_SCOPED_RANGE("setPaddingAndOffset");
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	int x64_sys = 0;

	// This function call returns 9999 for both major & minor fields, if no CUDA capable devices are present
	if (deviceProp.major == 9999 && deviceProp.minor == 9999){
		printf("Error: There is no device supporting CUDA.\n");
		exit(EXIT_FAILURE);
	}
    
    //check if double is used and supported
#ifdef _DOUBLE_SUPPORT_REQUIRED_
	printf("Simulation requires full precision double values\n");
	if ((deviceProp.major < 2)&&(deviceProp.minor < 3)){
		printf("Error: Hardware does not support full precision double values!\n");
		exit(EXIT_FAILURE);
	}
    
#endif

	//check 32 or 64bit
	x64_sys = (sizeof(void*)==8);
	if (x64_sys)
	{
		printf("64Bit System Detected\n");
	}
	else
	{
		printf("32Bit System Detected\n");
	}

	SM_START = 0;
	PADDING = 0;
  
	//copy padding and offset to GPU
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_SM_START), &SM_START, sizeof(int)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_PADDING), &PADDING, sizeof(int)));     
}

int is_sqr_pow2(int x){
	int r = (int)pow(4, ceil(log(x)/log(4)));
	return (r == x);
}

int lowest_sqr_pow2(int x){
	int l;
	
	//escape early if x is square power of 2
	if (is_sqr_pow2(x))
		return x;
	
	//lower bound		
	l = (int)pow(4, floor(log(x)/log(4)));
	
	return l;
}

/* Unary function required for hipOccupancyMaxPotentialBlockSizeVariableSMem to avoid warnings */
int no_sm(int b){
	return 0;
}

/* Unary function to return shared memory size for reorder message kernels */
int reorder_messages_sm_size(int blockSize)
{
	return sizeof(unsigned int)*(blockSize+1);
}


/** getIterationNumber
 *  Get the iteration number (host)
 *  @return a 1 indexed value for the iteration number, which is incremented at the start of each simulation step.
 *      I.e. it is 0 on up until the first call to singleIteration()
 */
extern unsigned int getIterationNumber(){
    return g_iterationNumber;
}

void initialise(char * inputfile){
    PROFILE_SCOPED_RANGE("initialise");

	//set the padding and offset values depending on architecture and OS
	setPaddingAndOffset();
  
    // Initialise some global variables
    g_iterationNumber = 0;

    // Initialise variables for tracking which iterations' data is accessible on the host.
    h_agents_default_variable_location_id_data_iteration = 0;
    h_agents_default_variable_agent_id_data_iteration = 0;
    h_agents_default_variable_state_data_iteration = 0;
    h_agents_default_variable_sugar_level_data_iteration = 0;
    h_agents_default_variable_metabolism_data_iteration = 0;
    h_agents_default_variable_env_sugar_level_data_iteration = 0;
    



	printf("Allocating Host and Device memory\n");
    PROFILE_PUSH_RANGE("allocate host");
	/* Agent memory allocation (CPU) */
	int xmachine_agent_SoA_size = sizeof(xmachine_memory_agent_list);
	h_agents_default = (xmachine_memory_agent_list*)malloc(xmachine_agent_SoA_size);

	/* Message memory allocation (CPU) */
	int message_cell_state_SoA_size = sizeof(xmachine_message_cell_state_list);
	h_cell_states = (xmachine_message_cell_state_list*)malloc(message_cell_state_SoA_size);
	int message_movement_request_SoA_size = sizeof(xmachine_message_movement_request_list);
	h_movement_requests = (xmachine_message_movement_request_list*)malloc(message_movement_request_SoA_size);
	int message_movement_response_SoA_size = sizeof(xmachine_message_movement_response_list);
	h_movement_responses = (xmachine_message_movement_response_list*)malloc(message_movement_response_SoA_size);

	//Exit if agent or message buffer sizes are to small for function outputs

  /* Graph memory allocation (CPU) */
  

    PROFILE_POP_RANGE(); //"allocate host"
	
	
	/* Set discrete cell_state message variables (range, width)*/
	h_message_cell_state_range = 1; //from xml
	h_message_cell_state_width = (int)floor(sqrt((float)xmachine_message_cell_state_MAX));
	//check the width
	if (!is_sqr_pow2(xmachine_message_cell_state_MAX)){
		printf("ERROR: cell_state message max must be a square power of 2 for a 2D discrete message grid!\n");
		exit(EXIT_FAILURE);
	}
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_cell_state_range), &h_message_cell_state_range, sizeof(int)));	
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_cell_state_width), &h_message_cell_state_width, sizeof(int)));
	
	
	/* Set discrete movement_request message variables (range, width)*/
	h_message_movement_request_range = 1; //from xml
	h_message_movement_request_width = (int)floor(sqrt((float)xmachine_message_movement_request_MAX));
	//check the width
	if (!is_sqr_pow2(xmachine_message_movement_request_MAX)){
		printf("ERROR: movement_request message max must be a square power of 2 for a 2D discrete message grid!\n");
		exit(EXIT_FAILURE);
	}
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_movement_request_range), &h_message_movement_request_range, sizeof(int)));	
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_movement_request_width), &h_message_movement_request_width, sizeof(int)));
	
	
	/* Set discrete movement_response message variables (range, width)*/
	h_message_movement_response_range = 1; //from xml
	h_message_movement_response_width = (int)floor(sqrt((float)xmachine_message_movement_response_MAX));
	//check the width
	if (!is_sqr_pow2(xmachine_message_movement_response_MAX)){
		printf("ERROR: movement_response message max must be a square power of 2 for a 2D discrete message grid!\n");
		exit(EXIT_FAILURE);
	}
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_movement_response_range), &h_message_movement_response_range, sizeof(int)));	
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_movement_response_width), &h_message_movement_response_width, sizeof(int)));
	
	/* Check that population size is a square power of 2*/
	if (!is_sqr_pow2(xmachine_memory_agent_MAX)){
		printf("ERROR: agents agent count must be a square power of 2!\n");
		exit(EXIT_FAILURE);
	}
	h_xmachine_memory_agent_pop_width = (int)sqrt(xmachine_memory_agent_MAX);
	

	//read initial states
	readInitialStates(inputfile, h_agents_default, &h_xmachine_memory_agent_default_count);

  // Read graphs from disk
  

  PROFILE_PUSH_RANGE("allocate device");
	
	/* agent Agent memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_agents, xmachine_agent_SoA_size));
	gpuErrchk( hipMalloc( (void**) &d_agents_swap, xmachine_agent_SoA_size));
	gpuErrchk( hipMalloc( (void**) &d_agents_new, xmachine_agent_SoA_size));
    
	/* default memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_agents_default, xmachine_agent_SoA_size));
	gpuErrchk( hipMemcpy( d_agents_default, h_agents_default, xmachine_agent_SoA_size, hipMemcpyHostToDevice));
    
	/* cell_state Message memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_cell_states, message_cell_state_SoA_size));
	gpuErrchk( hipMalloc( (void**) &d_cell_states_swap, message_cell_state_SoA_size));
	gpuErrchk( hipMemcpy( d_cell_states, h_cell_states, message_cell_state_SoA_size, hipMemcpyHostToDevice));
	
	/* movement_request Message memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_movement_requests, message_movement_request_SoA_size));
	gpuErrchk( hipMalloc( (void**) &d_movement_requests_swap, message_movement_request_SoA_size));
	gpuErrchk( hipMemcpy( d_movement_requests, h_movement_requests, message_movement_request_SoA_size, hipMemcpyHostToDevice));
	
	/* movement_response Message memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_movement_responses, message_movement_response_SoA_size));
	gpuErrchk( hipMalloc( (void**) &d_movement_responses_swap, message_movement_response_SoA_size));
	gpuErrchk( hipMemcpy( d_movement_responses, h_movement_responses, message_movement_response_SoA_size, hipMemcpyHostToDevice));
		


  /* Allocate device memory for graphs */
  

    PROFILE_POP_RANGE(); // "allocate device"

    /* Calculate and allocate CUB temporary memory for exclusive scans */
    
    d_temp_scan_storage_agent = nullptr;
    temp_scan_storage_bytes_agent = 0;
    hipcub::DeviceScan::ExclusiveSum(
        d_temp_scan_storage_agent, 
        temp_scan_storage_bytes_agent, 
        (int*) nullptr, 
        (int*) nullptr, 
        xmachine_memory_agent_MAX
    );
    gpuErrchk(hipMalloc(&d_temp_scan_storage_agent, temp_scan_storage_bytes_agent));
    

	/*Set global condition counts*/

	/* RNG rand48 */
    PROFILE_PUSH_RANGE("Initialse RNG_rand48");
	int h_rand48_SoA_size = sizeof(RNG_rand48);
	h_rand48 = (RNG_rand48*)malloc(h_rand48_SoA_size);
	//allocate on GPU
	gpuErrchk( hipMalloc( (void**) &d_rand48, h_rand48_SoA_size));
	// calculate strided iteration constants
	static const unsigned long long a = 0x5DEECE66DLL, c = 0xB;
	int seed = 123;
	unsigned long long A, C;
	A = 1LL; C = 0LL;
	for (unsigned int i = 0; i < buffer_size_MAX; ++i) {
		C += A*c;
		A *= a;
	}
	h_rand48->A.x = A & 0xFFFFFFLL;
	h_rand48->A.y = (A >> 24) & 0xFFFFFFLL;
	h_rand48->C.x = C & 0xFFFFFFLL;
	h_rand48->C.y = (C >> 24) & 0xFFFFFFLL;
	// prepare first nThreads random numbers from seed
	unsigned long long x = (((unsigned long long)seed) << 16) | 0x330E;
	for (unsigned int i = 0; i < buffer_size_MAX; ++i) {
		x = a*x + c;
		h_rand48->seeds[i].x = x & 0xFFFFFFLL;
		h_rand48->seeds[i].y = (x >> 24) & 0xFFFFFFLL;
	}
	//copy to device
	gpuErrchk( hipMemcpy( d_rand48, h_rand48, h_rand48_SoA_size, hipMemcpyHostToDevice));

    PROFILE_POP_RANGE();

	/* Call all init functions */
	/* Prepare cuda event timers for instrumentation */
#if defined(INSTRUMENT_ITERATIONS) && INSTRUMENT_ITERATIONS
	hipEventCreate(&instrument_iteration_start);
	hipEventCreate(&instrument_iteration_stop);
#endif
#if (defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS) || (defined(INSTRUMENT_INIT_FUNCTIONS) && INSTRUMENT_INIT_FUNCTIONS) || (defined(INSTRUMENT_STEP_FUNCTIONS) && INSTRUMENT_STEP_FUNCTIONS) || (defined(INSTRUMENT_EXIT_FUNCTIONS) && INSTRUMENT_EXIT_FUNCTIONS)
	hipEventCreate(&instrument_start);
	hipEventCreate(&instrument_stop);
#endif

	
  
  /* Init CUDA Streams for function layers */
  
  gpuErrchk(hipStreamCreate(&stream1));

#if defined(OUTPUT_POPULATION_PER_ITERATION) && OUTPUT_POPULATION_PER_ITERATION
	// Print the agent population size of all agents in all states
	
		printf("Init agent_agent_default_count: %u\n",get_agent_agent_default_count());
	
#endif
} 



void cleanup(){
    PROFILE_SCOPED_RANGE("cleanup");

    /* Call all exit functions */
	

	/* Agent data free*/
	
	/* agent Agent variables */
	gpuErrchk(hipFree(d_agents));
	gpuErrchk(hipFree(d_agents_swap));
	gpuErrchk(hipFree(d_agents_new));
	
	free( h_agents_default);
	gpuErrchk(hipFree(d_agents_default));
	

	/* Message data free */
	
	/* cell_state Message variables */
	free( h_cell_states);
	gpuErrchk(hipFree(d_cell_states));
	gpuErrchk(hipFree(d_cell_states_swap));
	
	/* movement_request Message variables */
	free( h_movement_requests);
	gpuErrchk(hipFree(d_movement_requests));
	gpuErrchk(hipFree(d_movement_requests_swap));
	
	/* movement_response Message variables */
	free( h_movement_responses);
	gpuErrchk(hipFree(d_movement_responses));
	gpuErrchk(hipFree(d_movement_responses_swap));
	

    /* Free temporary CUB memory if required. */
    
    if(d_temp_scan_storage_agent != nullptr){
      gpuErrchk(hipFree(d_temp_scan_storage_agent));
      d_temp_scan_storage_agent = nullptr;
      temp_scan_storage_bytes_agent = 0;
    }
    

  /* Graph data free */
  
  
  /* CUDA Streams for function layers */
  
  gpuErrchk(hipStreamDestroy(stream1));

  /* CUDA Event Timers for Instrumentation */
#if defined(INSTRUMENT_ITERATIONS) && INSTRUMENT_ITERATIONS
	hipEventDestroy(instrument_iteration_start);
	hipEventDestroy(instrument_iteration_stop);
#endif
#if (defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS) || (defined(INSTRUMENT_INIT_FUNCTIONS) && INSTRUMENT_INIT_FUNCTIONS) || (defined(INSTRUMENT_STEP_FUNCTIONS) && INSTRUMENT_STEP_FUNCTIONS) || (defined(INSTRUMENT_EXIT_FUNCTIONS) && INSTRUMENT_EXIT_FUNCTIONS)
	hipEventDestroy(instrument_start);
	hipEventDestroy(instrument_stop);
#endif
}

void singleIteration(){
PROFILE_SCOPED_RANGE("singleIteration");

#if defined(INSTRUMENT_ITERATIONS) && INSTRUMENT_ITERATIONS
	hipEventRecord(instrument_iteration_start);
#endif

    // Increment the iteration number.
    g_iterationNumber++;

  /* set all non partitioned, spatial partitioned and On-Graph Partitioned message counts to 0*/

	/* Call agent functions in order iterating through the layer functions */
	
	/* Layer 1*/
	
#if defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS
	hipEventRecord(instrument_start);
#endif
	
    PROFILE_PUSH_RANGE("agent_metabolise_and_growback");
	agent_metabolise_and_growback(stream1);
    PROFILE_POP_RANGE();
#if defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS
	hipEventRecord(instrument_stop);
	hipEventSynchronize(instrument_stop);
	hipEventElapsedTime(&instrument_milliseconds, instrument_start, instrument_stop);
	printf("Instrumentation: agent_metabolise_and_growback = %f (ms)\n", instrument_milliseconds);
#endif
	hipDeviceSynchronize();
  
	/* Layer 2*/
	
#if defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS
	hipEventRecord(instrument_start);
#endif
	
    PROFILE_PUSH_RANGE("agent_output_cell_state");
	agent_output_cell_state(stream1);
    PROFILE_POP_RANGE();
#if defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS
	hipEventRecord(instrument_stop);
	hipEventSynchronize(instrument_stop);
	hipEventElapsedTime(&instrument_milliseconds, instrument_start, instrument_stop);
	printf("Instrumentation: agent_output_cell_state = %f (ms)\n", instrument_milliseconds);
#endif
	hipDeviceSynchronize();
  
	/* Layer 3*/
	
#if defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS
	hipEventRecord(instrument_start);
#endif
	
    PROFILE_PUSH_RANGE("agent_movement_request");
	agent_movement_request(stream1);
    PROFILE_POP_RANGE();
#if defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS
	hipEventRecord(instrument_stop);
	hipEventSynchronize(instrument_stop);
	hipEventElapsedTime(&instrument_milliseconds, instrument_start, instrument_stop);
	printf("Instrumentation: agent_movement_request = %f (ms)\n", instrument_milliseconds);
#endif
	hipDeviceSynchronize();
  
	/* Layer 4*/
	
#if defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS
	hipEventRecord(instrument_start);
#endif
	
    PROFILE_PUSH_RANGE("agent_movement_response");
	agent_movement_response(stream1);
    PROFILE_POP_RANGE();
#if defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS
	hipEventRecord(instrument_stop);
	hipEventSynchronize(instrument_stop);
	hipEventElapsedTime(&instrument_milliseconds, instrument_start, instrument_stop);
	printf("Instrumentation: agent_movement_response = %f (ms)\n", instrument_milliseconds);
#endif
	hipDeviceSynchronize();
  
	/* Layer 5*/
	
#if defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS
	hipEventRecord(instrument_start);
#endif
	
    PROFILE_PUSH_RANGE("agent_movement_transaction");
	agent_movement_transaction(stream1);
    PROFILE_POP_RANGE();
#if defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS
	hipEventRecord(instrument_stop);
	hipEventSynchronize(instrument_stop);
	hipEventElapsedTime(&instrument_milliseconds, instrument_start, instrument_stop);
	printf("Instrumentation: agent_movement_transaction = %f (ms)\n", instrument_milliseconds);
#endif
	hipDeviceSynchronize();
  
    
    /* Call all step functions */
	

#if defined(OUTPUT_POPULATION_PER_ITERATION) && OUTPUT_POPULATION_PER_ITERATION
	// Print the agent population size of all agents in all states
	
		printf("agent_agent_default_count: %u\n",get_agent_agent_default_count());
	
#endif

#if defined(INSTRUMENT_ITERATIONS) && INSTRUMENT_ITERATIONS
	hipEventRecord(instrument_iteration_stop);
	hipEventSynchronize(instrument_iteration_stop);
	hipEventElapsedTime(&instrument_iteration_milliseconds, instrument_iteration_start, instrument_iteration_stop);
	printf("Instrumentation: Iteration Time = %f (ms)\n", instrument_iteration_milliseconds);
#endif
}

/* Environment functions */

//host constant declaration



/* Agent data access functions*/

    
int get_agent_agent_MAX_count(){
    return xmachine_memory_agent_MAX;
}


int get_agent_agent_default_count(){
	//discrete agent 
	return xmachine_memory_agent_MAX;
}

xmachine_memory_agent_list* get_device_agent_default_agents(){
	return d_agents_default;
}

xmachine_memory_agent_list* get_host_agent_default_agents(){
	return h_agents_default;
}

int get_agent_population_width(){
  return h_xmachine_memory_agent_pop_width;
}



/* Host based access of agent variables*/

/** int get_agent_default_variable_location_id(unsigned int index)
 * Gets the value of the location_id variable of an agent agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable location_id
 */
__host__ int get_agent_default_variable_location_id(unsigned int index){
    unsigned int count = get_agent_agent_default_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_agents_default_variable_location_id_data_iteration != currentIteration){
            gpuErrchk(
                hipMemcpy(
                    h_agents_default->location_id,
                    d_agents_default->location_id,
                    count * sizeof(int),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_agents_default_variable_location_id_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_agents_default->location_id[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access location_id for the %u th member of agent_default. count is %u at iteration %u\n", index, count, currentIteration);
        // Otherwise we return a default value
        return 0;

    }
}

/** int get_agent_default_variable_agent_id(unsigned int index)
 * Gets the value of the agent_id variable of an agent agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable agent_id
 */
__host__ int get_agent_default_variable_agent_id(unsigned int index){
    unsigned int count = get_agent_agent_default_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_agents_default_variable_agent_id_data_iteration != currentIteration){
            gpuErrchk(
                hipMemcpy(
                    h_agents_default->agent_id,
                    d_agents_default->agent_id,
                    count * sizeof(int),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_agents_default_variable_agent_id_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_agents_default->agent_id[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access agent_id for the %u th member of agent_default. count is %u at iteration %u\n", index, count, currentIteration);
        // Otherwise we return a default value
        return 0;

    }
}

/** int get_agent_default_variable_state(unsigned int index)
 * Gets the value of the state variable of an agent agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable state
 */
__host__ int get_agent_default_variable_state(unsigned int index){
    unsigned int count = get_agent_agent_default_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_agents_default_variable_state_data_iteration != currentIteration){
            gpuErrchk(
                hipMemcpy(
                    h_agents_default->state,
                    d_agents_default->state,
                    count * sizeof(int),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_agents_default_variable_state_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_agents_default->state[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access state for the %u th member of agent_default. count is %u at iteration %u\n", index, count, currentIteration);
        // Otherwise we return a default value
        return 0;

    }
}

/** int get_agent_default_variable_sugar_level(unsigned int index)
 * Gets the value of the sugar_level variable of an agent agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable sugar_level
 */
__host__ int get_agent_default_variable_sugar_level(unsigned int index){
    unsigned int count = get_agent_agent_default_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_agents_default_variable_sugar_level_data_iteration != currentIteration){
            gpuErrchk(
                hipMemcpy(
                    h_agents_default->sugar_level,
                    d_agents_default->sugar_level,
                    count * sizeof(int),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_agents_default_variable_sugar_level_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_agents_default->sugar_level[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access sugar_level for the %u th member of agent_default. count is %u at iteration %u\n", index, count, currentIteration);
        // Otherwise we return a default value
        return 0;

    }
}

/** int get_agent_default_variable_metabolism(unsigned int index)
 * Gets the value of the metabolism variable of an agent agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable metabolism
 */
__host__ int get_agent_default_variable_metabolism(unsigned int index){
    unsigned int count = get_agent_agent_default_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_agents_default_variable_metabolism_data_iteration != currentIteration){
            gpuErrchk(
                hipMemcpy(
                    h_agents_default->metabolism,
                    d_agents_default->metabolism,
                    count * sizeof(int),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_agents_default_variable_metabolism_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_agents_default->metabolism[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access metabolism for the %u th member of agent_default. count is %u at iteration %u\n", index, count, currentIteration);
        // Otherwise we return a default value
        return 0;

    }
}

/** int get_agent_default_variable_env_sugar_level(unsigned int index)
 * Gets the value of the env_sugar_level variable of an agent agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable env_sugar_level
 */
__host__ int get_agent_default_variable_env_sugar_level(unsigned int index){
    unsigned int count = get_agent_agent_default_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_agents_default_variable_env_sugar_level_data_iteration != currentIteration){
            gpuErrchk(
                hipMemcpy(
                    h_agents_default->env_sugar_level,
                    d_agents_default->env_sugar_level,
                    count * sizeof(int),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_agents_default_variable_env_sugar_level_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_agents_default->env_sugar_level[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access env_sugar_level for the %u th member of agent_default. count is %u at iteration %u\n", index, count, currentIteration);
        // Otherwise we return a default value
        return 0;

    }
}



/* Host based agent creation functions */
// These are only available for continuous agents.



/*  Analytics Functions */

int reduce_agent_default_location_id_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_agents_default->location_id),  thrust::device_pointer_cast(d_agents_default->location_id) + h_xmachine_memory_agent_default_count);
}

int count_agent_default_location_id_variable(int count_value){
    //count in default stream
    return (int)thrust::count(thrust::device_pointer_cast(d_agents_default->location_id),  thrust::device_pointer_cast(d_agents_default->location_id) + h_xmachine_memory_agent_default_count, count_value);
}
int min_agent_default_location_id_variable(){
    //min in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_agents_default->location_id);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_agent_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
int max_agent_default_location_id_variable(){
    //max in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_agents_default->location_id);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_agent_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
int reduce_agent_default_agent_id_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_agents_default->agent_id),  thrust::device_pointer_cast(d_agents_default->agent_id) + h_xmachine_memory_agent_default_count);
}

int count_agent_default_agent_id_variable(int count_value){
    //count in default stream
    return (int)thrust::count(thrust::device_pointer_cast(d_agents_default->agent_id),  thrust::device_pointer_cast(d_agents_default->agent_id) + h_xmachine_memory_agent_default_count, count_value);
}
int min_agent_default_agent_id_variable(){
    //min in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_agents_default->agent_id);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_agent_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
int max_agent_default_agent_id_variable(){
    //max in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_agents_default->agent_id);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_agent_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
int reduce_agent_default_state_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_agents_default->state),  thrust::device_pointer_cast(d_agents_default->state) + h_xmachine_memory_agent_default_count);
}

int count_agent_default_state_variable(int count_value){
    //count in default stream
    return (int)thrust::count(thrust::device_pointer_cast(d_agents_default->state),  thrust::device_pointer_cast(d_agents_default->state) + h_xmachine_memory_agent_default_count, count_value);
}
int min_agent_default_state_variable(){
    //min in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_agents_default->state);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_agent_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
int max_agent_default_state_variable(){
    //max in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_agents_default->state);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_agent_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
int reduce_agent_default_sugar_level_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_agents_default->sugar_level),  thrust::device_pointer_cast(d_agents_default->sugar_level) + h_xmachine_memory_agent_default_count);
}

int count_agent_default_sugar_level_variable(int count_value){
    //count in default stream
    return (int)thrust::count(thrust::device_pointer_cast(d_agents_default->sugar_level),  thrust::device_pointer_cast(d_agents_default->sugar_level) + h_xmachine_memory_agent_default_count, count_value);
}
int min_agent_default_sugar_level_variable(){
    //min in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_agents_default->sugar_level);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_agent_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
int max_agent_default_sugar_level_variable(){
    //max in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_agents_default->sugar_level);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_agent_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
int reduce_agent_default_metabolism_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_agents_default->metabolism),  thrust::device_pointer_cast(d_agents_default->metabolism) + h_xmachine_memory_agent_default_count);
}

int count_agent_default_metabolism_variable(int count_value){
    //count in default stream
    return (int)thrust::count(thrust::device_pointer_cast(d_agents_default->metabolism),  thrust::device_pointer_cast(d_agents_default->metabolism) + h_xmachine_memory_agent_default_count, count_value);
}
int min_agent_default_metabolism_variable(){
    //min in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_agents_default->metabolism);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_agent_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
int max_agent_default_metabolism_variable(){
    //max in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_agents_default->metabolism);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_agent_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
int reduce_agent_default_env_sugar_level_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_agents_default->env_sugar_level),  thrust::device_pointer_cast(d_agents_default->env_sugar_level) + h_xmachine_memory_agent_default_count);
}

int count_agent_default_env_sugar_level_variable(int count_value){
    //count in default stream
    return (int)thrust::count(thrust::device_pointer_cast(d_agents_default->env_sugar_level),  thrust::device_pointer_cast(d_agents_default->env_sugar_level) + h_xmachine_memory_agent_default_count, count_value);
}
int min_agent_default_env_sugar_level_variable(){
    //min in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_agents_default->env_sugar_level);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_agent_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
int max_agent_default_env_sugar_level_variable(){
    //max in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_agents_default->env_sugar_level);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_agent_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}



/* Agent functions */


	
/* Shared memory size calculator for agent function */
int agent_metabolise_and_growback_sm_size(int blockSize){
	int sm_size;
	sm_size = SM_START;
  
	return sm_size;
}

/** agent_metabolise_and_growback
 * Agent function prototype for metabolise_and_growback function of agent agent
 */
void agent_metabolise_and_growback(hipStream_t &stream){

    int sm_size;
    int blockSize;
    int minGridSize;
    int gridSize;
    int state_list_size;
	dim3 g; //grid for agent func
	dim3 b; //block for agent func

	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_agent_default_count == 0)
	{
		return;
	}
	
	
	//SET SM size to 0 and save state list size for occupancy calculations
	sm_size = SM_START;
	state_list_size = h_xmachine_memory_agent_default_count;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS A GLOBAL CONDITION
	
	//COPY CURRENT STATE COUNT TO WORKING COUNT (host and device)
	h_xmachine_memory_agent_count = h_xmachine_memory_agent_default_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_count), &h_xmachine_memory_agent_count, sizeof(int)));	
	
	//RESET SCAN INPUTS
	//reset scan input for currentState
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, reset_agent_scan_input, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	reset_agent_scan_input<<<gridSize, blockSize, 0, stream>>>(d_agents_default);
	gpuErrchkLaunch();
	
	//APPLY FUNCTION FILTER
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, metabolise_and_growback_function_filter, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	metabolise_and_growback_function_filter<<<gridSize, blockSize, 0, stream>>>(d_agents_default);
	gpuErrchkLaunch();
	
	//GET CONDTIONS TRUE COUNT FROM CURRENT STATE LIST
    hipcub::DeviceScan::ExclusiveSum(
        d_temp_scan_storage_agent, 
        temp_scan_storage_bytes_agent, 
        d_agents_default->_scan_input,
        d_agents_default->_position,
        h_xmachine_memory_agent_count, 
        stream
    );

	//reset agent count
	gpuErrchk( hipMemcpy( &scan_last_sum, &d_agents_default->_position[h_xmachine_memory_agent_count-1], sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk( hipMemcpy( &scan_last_included, &d_agents_default->_scan_input[h_xmachine_memory_agent_count-1], sizeof(int), hipMemcpyDeviceToHost));
	int global_conditions_true = 0;
	if (scan_last_included == 1)
		global_conditions_true = scan_last_sum+1;
	else		
		global_conditions_true = scan_last_sum;
	//check if condition is true for all agents or if max condition count is reached
	if ((global_conditions_true != h_xmachine_memory_agent_count)&&(h_metabolise_and_growback_condition_count < 9))
	{
		h_metabolise_and_growback_condition_count ++;
		return;
	}
	if ((h_metabolise_and_growback_condition_count == 9))
	{
		printf("Global agent condition for metabolise_and_growback function reached the maximum number of 9 conditions\n");
	}
	
	//RESET THE CONDITION COUNT
	h_metabolise_and_growback_condition_count = 0;
	
	//MAP CURRENT STATE TO WORKING LIST
	xmachine_memory_agent_list* agents_default_temp = d_agents;
	d_agents = d_agents_default;
	d_agents_default = agents_default_temp;
	//set current state count to 0
	h_xmachine_memory_agent_default_count = 0;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_count), &h_xmachine_memory_agent_count, sizeof(int)));	
	
	
	
 

	//******************************** AGENT FUNCTION *******************************

	
	
	//calculate the grid block size for main agent function
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, GPUFLAME_metabolise_and_growback, agent_metabolise_and_growback_sm_size, state_list_size);
	blockSize = lowest_sqr_pow2(blockSize); //For discrete agents the block size must be a square power of 2
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	b.x = (int)sqrt(blockSize);
	b.y = b.x;
	g.x = (int)sqrt(gridSize);
	g.y = g.x;
	sm_size = agent_metabolise_and_growback_sm_size(blockSize);
	
	
	
	
	//MAIN XMACHINE FUNCTION CALL (metabolise_and_growback)
	//Reallocate   : false
	//Input        : 
	//Output       : 
	//Agent Output : 
	GPUFLAME_metabolise_and_growback<<<g, b, sm_size, stream>>>(d_agents);
	gpuErrchkLaunch();
	
	
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
    //currentState maps to working list
	agents_default_temp = d_agents_default;
	d_agents_default = d_agents;
	d_agents = agents_default_temp;
    //set current state count
	h_xmachine_memory_agent_default_count = h_xmachine_memory_agent_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_default_count), &h_xmachine_memory_agent_default_count, sizeof(int)));	
	
	
}



	
/* Shared memory size calculator for agent function */
int agent_output_cell_state_sm_size(int blockSize){
	int sm_size;
	sm_size = SM_START;
  
	return sm_size;
}

/** agent_output_cell_state
 * Agent function prototype for output_cell_state function of agent agent
 */
void agent_output_cell_state(hipStream_t &stream){

    int sm_size;
    int blockSize;
    int minGridSize;
    int gridSize;
    int state_list_size;
	dim3 g; //grid for agent func
	dim3 b; //block for agent func

	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_agent_default_count == 0)
	{
		return;
	}
	
	
	//SET SM size to 0 and save state list size for occupancy calculations
	sm_size = SM_START;
	state_list_size = h_xmachine_memory_agent_default_count;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS NOT A FUNCTION CONDITION
	//currentState maps to working list
	xmachine_memory_agent_list* agents_default_temp = d_agents;
	d_agents = d_agents_default;
	d_agents_default = agents_default_temp;
	//set working count to current state count
	h_xmachine_memory_agent_count = h_xmachine_memory_agent_default_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_count), &h_xmachine_memory_agent_count, sizeof(int)));	
	//set current state count to 0
	h_xmachine_memory_agent_default_count = 0;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_default_count), &h_xmachine_memory_agent_default_count, sizeof(int)));	
	
 

	//******************************** AGENT FUNCTION *******************************

	
	
	//calculate the grid block size for main agent function
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, GPUFLAME_output_cell_state, agent_output_cell_state_sm_size, state_list_size);
	blockSize = lowest_sqr_pow2(blockSize); //For discrete agents the block size must be a square power of 2
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	b.x = (int)sqrt(blockSize);
	b.y = b.x;
	g.x = (int)sqrt(gridSize);
	g.y = g.x;
	sm_size = agent_output_cell_state_sm_size(blockSize);
	
	
	
	//SET THE OUTPUT MESSAGE TYPE FOR CONTINUOUS AGENTS
	
	
	//MAIN XMACHINE FUNCTION CALL (output_cell_state)
	//Reallocate   : false
	//Input        : 
	//Output       : cell_state
	//Agent Output : 
	GPUFLAME_output_cell_state<<<g, b, sm_size, stream>>>(d_agents, d_cell_states);
	gpuErrchkLaunch();
	
	
	//CONTINUOUS AGENTS SCATTER NON PARTITIONED OPTIONAL OUTPUT MESSAGES
	
	//UPDATE MESSAGE COUNTS FOR CONTINUOUS AGENTS WITH NON PARTITIONED MESSAGE OUTPUT 
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
    //currentState maps to working list
	agents_default_temp = d_agents_default;
	d_agents_default = d_agents;
	d_agents = agents_default_temp;
    //set current state count
	h_xmachine_memory_agent_default_count = h_xmachine_memory_agent_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_default_count), &h_xmachine_memory_agent_default_count, sizeof(int)));	
	
	
}



	
/* Shared memory size calculator for agent function */
int agent_movement_request_sm_size(int blockSize){
	int sm_size;
	sm_size = SM_START;
  //Discrete agent and message input has discrete partitioning
	int sm_grid_width = (int)ceil(sqrt(blockSize));
	int sm_grid_size = (int)pow((float)sm_grid_width+(h_message_cell_state_range*2), 2);
	sm_size += (sm_grid_size *sizeof(xmachine_message_cell_state)); //update sm size
	sm_size += (sm_grid_size * PADDING);  //offset for avoiding conflicts
	
	return sm_size;
}

/** agent_movement_request
 * Agent function prototype for movement_request function of agent agent
 */
void agent_movement_request(hipStream_t &stream){

    int sm_size;
    int blockSize;
    int minGridSize;
    int gridSize;
    int state_list_size;
	dim3 g; //grid for agent func
	dim3 b; //block for agent func

	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_agent_default_count == 0)
	{
		return;
	}
	
	
	//SET SM size to 0 and save state list size for occupancy calculations
	sm_size = SM_START;
	state_list_size = h_xmachine_memory_agent_default_count;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS NOT A FUNCTION CONDITION
	//currentState maps to working list
	xmachine_memory_agent_list* agents_default_temp = d_agents;
	d_agents = d_agents_default;
	d_agents_default = agents_default_temp;
	//set working count to current state count
	h_xmachine_memory_agent_count = h_xmachine_memory_agent_default_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_count), &h_xmachine_memory_agent_count, sizeof(int)));	
	//set current state count to 0
	h_xmachine_memory_agent_default_count = 0;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_default_count), &h_xmachine_memory_agent_default_count, sizeof(int)));	
	
 

	//******************************** AGENT FUNCTION *******************************

	
	
	//calculate the grid block size for main agent function
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, GPUFLAME_movement_request, agent_movement_request_sm_size, state_list_size);
	blockSize = lowest_sqr_pow2(blockSize); //For discrete agents the block size must be a square power of 2
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	b.x = (int)sqrt(blockSize);
	b.y = b.x;
	g.x = (int)sqrt(gridSize);
	g.y = g.x;
	sm_size = agent_movement_request_sm_size(blockSize);
	
	
	
	//check that the range is not greater than the square of the block size. If so then there will be too many uncoalesded reads
	if (h_message_cell_state_range > (int)blockSize){
		printf("ERROR: Message range is greater than the thread block size. Increase thread block size or reduce the range!\n");
		exit(EXIT_FAILURE);
	}
	
	//BIND APPROPRIATE MESSAGE INPUT VARIABLES TO TEXTURES (to make use of the texture cache)
	//any agent with discrete or partitioned message input uses texture caching
	size_t tex_xmachine_message_cell_state_location_id_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_cell_state_location_id_byte_offset, tex_xmachine_message_cell_state_location_id, d_cell_states->location_id, sizeof(int)*xmachine_message_cell_state_MAX));
	h_tex_xmachine_message_cell_state_location_id_offset = (int)tex_xmachine_message_cell_state_location_id_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_cell_state_location_id_offset), &h_tex_xmachine_message_cell_state_location_id_offset, sizeof(int)));
	size_t tex_xmachine_message_cell_state_state_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_cell_state_state_byte_offset, tex_xmachine_message_cell_state_state, d_cell_states->state, sizeof(int)*xmachine_message_cell_state_MAX));
	h_tex_xmachine_message_cell_state_state_offset = (int)tex_xmachine_message_cell_state_state_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_cell_state_state_offset), &h_tex_xmachine_message_cell_state_state_offset, sizeof(int)));
	size_t tex_xmachine_message_cell_state_env_sugar_level_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_cell_state_env_sugar_level_byte_offset, tex_xmachine_message_cell_state_env_sugar_level, d_cell_states->env_sugar_level, sizeof(int)*xmachine_message_cell_state_MAX));
	h_tex_xmachine_message_cell_state_env_sugar_level_offset = (int)tex_xmachine_message_cell_state_env_sugar_level_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_cell_state_env_sugar_level_offset), &h_tex_xmachine_message_cell_state_env_sugar_level_offset, sizeof(int)));
	
	//SET THE OUTPUT MESSAGE TYPE FOR CONTINUOUS AGENTS
	
	
	//MAIN XMACHINE FUNCTION CALL (movement_request)
	//Reallocate   : false
	//Input        : cell_state
	//Output       : movement_request
	//Agent Output : 
	GPUFLAME_movement_request<<<g, b, sm_size, stream>>>(d_agents, d_cell_states, d_movement_requests);
	gpuErrchkLaunch();
	
	
	//UNBIND MESSAGE INPUT VARIABLE TEXTURES
	//any agent with discrete or partitioned message input uses texture caching
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_cell_state_location_id));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_cell_state_state));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_cell_state_env_sugar_level));
	
	//CONTINUOUS AGENTS SCATTER NON PARTITIONED OPTIONAL OUTPUT MESSAGES
	
	//UPDATE MESSAGE COUNTS FOR CONTINUOUS AGENTS WITH NON PARTITIONED MESSAGE OUTPUT 
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
    //currentState maps to working list
	agents_default_temp = d_agents_default;
	d_agents_default = d_agents;
	d_agents = agents_default_temp;
    //set current state count
	h_xmachine_memory_agent_default_count = h_xmachine_memory_agent_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_default_count), &h_xmachine_memory_agent_default_count, sizeof(int)));	
	
	
}



	
/* Shared memory size calculator for agent function */
int agent_movement_response_sm_size(int blockSize){
	int sm_size;
	sm_size = SM_START;
  //Discrete agent and message input has discrete partitioning
	int sm_grid_width = (int)ceil(sqrt(blockSize));
	int sm_grid_size = (int)pow((float)sm_grid_width+(h_message_movement_request_range*2), 2);
	sm_size += (sm_grid_size *sizeof(xmachine_message_movement_request)); //update sm size
	sm_size += (sm_grid_size * PADDING);  //offset for avoiding conflicts
	
	return sm_size;
}

/** agent_movement_response
 * Agent function prototype for movement_response function of agent agent
 */
void agent_movement_response(hipStream_t &stream){

    int sm_size;
    int blockSize;
    int minGridSize;
    int gridSize;
    int state_list_size;
	dim3 g; //grid for agent func
	dim3 b; //block for agent func

	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_agent_default_count == 0)
	{
		return;
	}
	
	
	//SET SM size to 0 and save state list size for occupancy calculations
	sm_size = SM_START;
	state_list_size = h_xmachine_memory_agent_default_count;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS NOT A FUNCTION CONDITION
	//currentState maps to working list
	xmachine_memory_agent_list* agents_default_temp = d_agents;
	d_agents = d_agents_default;
	d_agents_default = agents_default_temp;
	//set working count to current state count
	h_xmachine_memory_agent_count = h_xmachine_memory_agent_default_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_count), &h_xmachine_memory_agent_count, sizeof(int)));	
	//set current state count to 0
	h_xmachine_memory_agent_default_count = 0;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_default_count), &h_xmachine_memory_agent_default_count, sizeof(int)));	
	
 

	//******************************** AGENT FUNCTION *******************************

	
	
	//calculate the grid block size for main agent function
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, GPUFLAME_movement_response, agent_movement_response_sm_size, state_list_size);
	blockSize = lowest_sqr_pow2(blockSize); //For discrete agents the block size must be a square power of 2
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	b.x = (int)sqrt(blockSize);
	b.y = b.x;
	g.x = (int)sqrt(gridSize);
	g.y = g.x;
	sm_size = agent_movement_response_sm_size(blockSize);
	
	
	
	//check that the range is not greater than the square of the block size. If so then there will be too many uncoalesded reads
	if (h_message_movement_request_range > (int)blockSize){
		printf("ERROR: Message range is greater than the thread block size. Increase thread block size or reduce the range!\n");
		exit(EXIT_FAILURE);
	}
	
	//BIND APPROPRIATE MESSAGE INPUT VARIABLES TO TEXTURES (to make use of the texture cache)
	//any agent with discrete or partitioned message input uses texture caching
	size_t tex_xmachine_message_movement_request_agent_id_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_movement_request_agent_id_byte_offset, tex_xmachine_message_movement_request_agent_id, d_movement_requests->agent_id, sizeof(int)*xmachine_message_movement_request_MAX));
	h_tex_xmachine_message_movement_request_agent_id_offset = (int)tex_xmachine_message_movement_request_agent_id_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_movement_request_agent_id_offset), &h_tex_xmachine_message_movement_request_agent_id_offset, sizeof(int)));
	size_t tex_xmachine_message_movement_request_location_id_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_movement_request_location_id_byte_offset, tex_xmachine_message_movement_request_location_id, d_movement_requests->location_id, sizeof(int)*xmachine_message_movement_request_MAX));
	h_tex_xmachine_message_movement_request_location_id_offset = (int)tex_xmachine_message_movement_request_location_id_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_movement_request_location_id_offset), &h_tex_xmachine_message_movement_request_location_id_offset, sizeof(int)));
	size_t tex_xmachine_message_movement_request_sugar_level_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_movement_request_sugar_level_byte_offset, tex_xmachine_message_movement_request_sugar_level, d_movement_requests->sugar_level, sizeof(int)*xmachine_message_movement_request_MAX));
	h_tex_xmachine_message_movement_request_sugar_level_offset = (int)tex_xmachine_message_movement_request_sugar_level_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_movement_request_sugar_level_offset), &h_tex_xmachine_message_movement_request_sugar_level_offset, sizeof(int)));
	size_t tex_xmachine_message_movement_request_metabolism_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_movement_request_metabolism_byte_offset, tex_xmachine_message_movement_request_metabolism, d_movement_requests->metabolism, sizeof(int)*xmachine_message_movement_request_MAX));
	h_tex_xmachine_message_movement_request_metabolism_offset = (int)tex_xmachine_message_movement_request_metabolism_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_movement_request_metabolism_offset), &h_tex_xmachine_message_movement_request_metabolism_offset, sizeof(int)));
	
	//SET THE OUTPUT MESSAGE TYPE FOR CONTINUOUS AGENTS
	
	
	//MAIN XMACHINE FUNCTION CALL (movement_response)
	//Reallocate   : false
	//Input        : movement_request
	//Output       : movement_response
	//Agent Output : 
	GPUFLAME_movement_response<<<g, b, sm_size, stream>>>(d_agents, d_movement_requests, d_movement_responses, d_rand48);
	gpuErrchkLaunch();
	
	
	//UNBIND MESSAGE INPUT VARIABLE TEXTURES
	//any agent with discrete or partitioned message input uses texture caching
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_movement_request_agent_id));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_movement_request_location_id));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_movement_request_sugar_level));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_movement_request_metabolism));
	
	//CONTINUOUS AGENTS SCATTER NON PARTITIONED OPTIONAL OUTPUT MESSAGES
	
	//UPDATE MESSAGE COUNTS FOR CONTINUOUS AGENTS WITH NON PARTITIONED MESSAGE OUTPUT 
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
    //currentState maps to working list
	agents_default_temp = d_agents_default;
	d_agents_default = d_agents;
	d_agents = agents_default_temp;
    //set current state count
	h_xmachine_memory_agent_default_count = h_xmachine_memory_agent_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_default_count), &h_xmachine_memory_agent_default_count, sizeof(int)));	
	
	
}



	
/* Shared memory size calculator for agent function */
int agent_movement_transaction_sm_size(int blockSize){
	int sm_size;
	sm_size = SM_START;
  //Discrete agent and message input has discrete partitioning
	int sm_grid_width = (int)ceil(sqrt(blockSize));
	int sm_grid_size = (int)pow((float)sm_grid_width+(h_message_movement_response_range*2), 2);
	sm_size += (sm_grid_size *sizeof(xmachine_message_movement_response)); //update sm size
	sm_size += (sm_grid_size * PADDING);  //offset for avoiding conflicts
	
	return sm_size;
}

/** agent_movement_transaction
 * Agent function prototype for movement_transaction function of agent agent
 */
void agent_movement_transaction(hipStream_t &stream){

    int sm_size;
    int blockSize;
    int minGridSize;
    int gridSize;
    int state_list_size;
	dim3 g; //grid for agent func
	dim3 b; //block for agent func

	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_agent_default_count == 0)
	{
		return;
	}
	
	
	//SET SM size to 0 and save state list size for occupancy calculations
	sm_size = SM_START;
	state_list_size = h_xmachine_memory_agent_default_count;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS NOT A FUNCTION CONDITION
	//currentState maps to working list
	xmachine_memory_agent_list* agents_default_temp = d_agents;
	d_agents = d_agents_default;
	d_agents_default = agents_default_temp;
	//set working count to current state count
	h_xmachine_memory_agent_count = h_xmachine_memory_agent_default_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_count), &h_xmachine_memory_agent_count, sizeof(int)));	
	//set current state count to 0
	h_xmachine_memory_agent_default_count = 0;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_default_count), &h_xmachine_memory_agent_default_count, sizeof(int)));	
	
 

	//******************************** AGENT FUNCTION *******************************

	
	
	//calculate the grid block size for main agent function
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, GPUFLAME_movement_transaction, agent_movement_transaction_sm_size, state_list_size);
	blockSize = lowest_sqr_pow2(blockSize); //For discrete agents the block size must be a square power of 2
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	b.x = (int)sqrt(blockSize);
	b.y = b.x;
	g.x = (int)sqrt(gridSize);
	g.y = g.x;
	sm_size = agent_movement_transaction_sm_size(blockSize);
	
	
	
	//check that the range is not greater than the square of the block size. If so then there will be too many uncoalesded reads
	if (h_message_movement_response_range > (int)blockSize){
		printf("ERROR: Message range is greater than the thread block size. Increase thread block size or reduce the range!\n");
		exit(EXIT_FAILURE);
	}
	
	//BIND APPROPRIATE MESSAGE INPUT VARIABLES TO TEXTURES (to make use of the texture cache)
	//any agent with discrete or partitioned message input uses texture caching
	size_t tex_xmachine_message_movement_response_location_id_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_movement_response_location_id_byte_offset, tex_xmachine_message_movement_response_location_id, d_movement_responses->location_id, sizeof(int)*xmachine_message_movement_response_MAX));
	h_tex_xmachine_message_movement_response_location_id_offset = (int)tex_xmachine_message_movement_response_location_id_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_movement_response_location_id_offset), &h_tex_xmachine_message_movement_response_location_id_offset, sizeof(int)));
	size_t tex_xmachine_message_movement_response_agent_id_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_movement_response_agent_id_byte_offset, tex_xmachine_message_movement_response_agent_id, d_movement_responses->agent_id, sizeof(int)*xmachine_message_movement_response_MAX));
	h_tex_xmachine_message_movement_response_agent_id_offset = (int)tex_xmachine_message_movement_response_agent_id_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_movement_response_agent_id_offset), &h_tex_xmachine_message_movement_response_agent_id_offset, sizeof(int)));
	
	
	//MAIN XMACHINE FUNCTION CALL (movement_transaction)
	//Reallocate   : false
	//Input        : movement_response
	//Output       : 
	//Agent Output : 
	GPUFLAME_movement_transaction<<<g, b, sm_size, stream>>>(d_agents, d_movement_responses);
	gpuErrchkLaunch();
	
	
	//UNBIND MESSAGE INPUT VARIABLE TEXTURES
	//any agent with discrete or partitioned message input uses texture caching
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_movement_response_location_id));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_movement_response_agent_id));
	
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
    //currentState maps to working list
	agents_default_temp = d_agents_default;
	d_agents_default = d_agents;
	d_agents = agents_default_temp;
    //set current state count
	h_xmachine_memory_agent_default_count = h_xmachine_memory_agent_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_agent_default_count), &h_xmachine_memory_agent_default_count, sizeof(int)));	
	
	
}


 
extern void reset_agent_default_count()
{
    h_xmachine_memory_agent_default_count = 0;
}
