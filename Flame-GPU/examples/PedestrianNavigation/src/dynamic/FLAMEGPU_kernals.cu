#include "hip/hip_runtime.h"

/*
 * FLAME GPU v 1.5.X for CUDA 9
 * Copyright University of Sheffield.
 * Original Author: Dr Paul Richmond (user contributions tracked on https://github.com/FLAMEGPU/FLAMEGPU)
 * Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
 *
 * University of Sheffield retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * University of Sheffield is strictly prohibited.
 *
 * For terms of licence agreement please attached licence or view licence
 * on www.flamegpu.com website.
 *
 */


#ifndef _FLAMEGPU_KERNELS_H_
#define _FLAMEGPU_KERNELS_H_

#include "header.h"


/* Agent count constants */

__constant__ int d_xmachine_memory_agent_count;

__constant__ int d_xmachine_memory_medic_count;

__constant__ int d_xmachine_memory_receptionist_count;

__constant__ int d_xmachine_memory_navmap_count;

/* Agent state count constants */

__constant__ int d_xmachine_memory_agent_default_count;

__constant__ int d_xmachine_memory_medic_default2_count;

__constant__ int d_xmachine_memory_receptionist_defaultReceptionist_count;

__constant__ int d_xmachine_memory_navmap_static_count;


/* Message constants */

/* pedestrian_location Message variables */
/* Non partitioned, spatial partitioned and on-graph partitioned message variables  */
__constant__ int d_message_pedestrian_location_count;         /**< message list counter*/
__constant__ int d_message_pedestrian_location_output_type;   /**< message output type (single or optional)*/
//Spatial Partitioning Variables
__constant__ glm::vec3 d_message_pedestrian_location_min_bounds;           /**< min bounds (x,y,z) of partitioning environment */
__constant__ glm::vec3 d_message_pedestrian_location_max_bounds;           /**< max bounds (x,y,z) of partitioning environment */
__constant__ glm::ivec3 d_message_pedestrian_location_partitionDim;           /**< partition dimensions (x,y,z) of partitioning environment */
__constant__ float d_message_pedestrian_location_radius;                 /**< partition radius (used to determin the size of the partitions) */

/* pedestrian_state Message variables */
/* Non partitioned, spatial partitioned and on-graph partitioned message variables  */
__constant__ int d_message_pedestrian_state_count;         /**< message list counter*/
__constant__ int d_message_pedestrian_state_output_type;   /**< message output type (single or optional)*/
//Spatial Partitioning Variables
__constant__ glm::vec3 d_message_pedestrian_state_min_bounds;           /**< min bounds (x,y,z) of partitioning environment */
__constant__ glm::vec3 d_message_pedestrian_state_max_bounds;           /**< max bounds (x,y,z) of partitioning environment */
__constant__ glm::ivec3 d_message_pedestrian_state_partitionDim;           /**< partition dimensions (x,y,z) of partitioning environment */
__constant__ float d_message_pedestrian_state_radius;                 /**< partition radius (used to determin the size of the partitions) */

/* check_in Message variables */
/* Non partitioned, spatial partitioned and on-graph partitioned message variables  */
__constant__ int d_message_check_in_count;         /**< message list counter*/
__constant__ int d_message_check_in_output_type;   /**< message output type (single or optional)*/

/* avisar_paciente Message variables */
/* Non partitioned, spatial partitioned and on-graph partitioned message variables  */
__constant__ int d_message_avisar_paciente_count;         /**< message list counter*/
__constant__ int d_message_avisar_paciente_output_type;   /**< message output type (single or optional)*/

/* navmap_cell Message variables */
//Discrete Partitioning Variables
__constant__ int d_message_navmap_cell_range;     /**< range of the discrete message*/
__constant__ int d_message_navmap_cell_width;     /**< with of the message grid*/

	

/* Graph Constants */


/* Graph device array pointer(s) */


/* Graph host array pointer(s) */

    
//include each function file

#include "functions.c"
    
/* Texture bindings */
/* pedestrian_location Message Bindings */texture<float, 1, hipReadModeElementType> tex_xmachine_message_pedestrian_location_x;
__constant__ int d_tex_xmachine_message_pedestrian_location_x_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_pedestrian_location_y;
__constant__ int d_tex_xmachine_message_pedestrian_location_y_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_pedestrian_location_z;
__constant__ int d_tex_xmachine_message_pedestrian_location_z_offset;texture<int, 1, hipReadModeElementType> tex_xmachine_message_pedestrian_location_estado;
__constant__ int d_tex_xmachine_message_pedestrian_location_estado_offset;
texture<int, 1, hipReadModeElementType> tex_xmachine_message_pedestrian_location_pbm_start;
__constant__ int d_tex_xmachine_message_pedestrian_location_pbm_start_offset;
texture<int, 1, hipReadModeElementType> tex_xmachine_message_pedestrian_location_pbm_end_or_count;
__constant__ int d_tex_xmachine_message_pedestrian_location_pbm_end_or_count_offset;


/* pedestrian_state Message Bindings */texture<float, 1, hipReadModeElementType> tex_xmachine_message_pedestrian_state_x;
__constant__ int d_tex_xmachine_message_pedestrian_state_x_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_pedestrian_state_y;
__constant__ int d_tex_xmachine_message_pedestrian_state_y_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_pedestrian_state_z;
__constant__ int d_tex_xmachine_message_pedestrian_state_z_offset;texture<int, 1, hipReadModeElementType> tex_xmachine_message_pedestrian_state_estado;
__constant__ int d_tex_xmachine_message_pedestrian_state_estado_offset;
texture<int, 1, hipReadModeElementType> tex_xmachine_message_pedestrian_state_pbm_start;
__constant__ int d_tex_xmachine_message_pedestrian_state_pbm_start_offset;
texture<int, 1, hipReadModeElementType> tex_xmachine_message_pedestrian_state_pbm_end_or_count;
__constant__ int d_tex_xmachine_message_pedestrian_state_pbm_end_or_count_offset;




/* navmap_cell Message Bindings */texture<int, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_x;
__constant__ int d_tex_xmachine_message_navmap_cell_x_offset;texture<int, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_y;
__constant__ int d_tex_xmachine_message_navmap_cell_y_offset;texture<int, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_exit_no;
__constant__ int d_tex_xmachine_message_navmap_cell_exit_no_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_height;
__constant__ int d_tex_xmachine_message_navmap_cell_height_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_collision_x;
__constant__ int d_tex_xmachine_message_navmap_cell_collision_x_offset;texture<float, 1, hipReadModeElementType> tex_xmachine_message_navmap_cell_collision_y;
__constant__ int d_tex_xmachine_message_navmap_cell_collision_y_offset;

    
#define WRAP(x,m) (((x)<m)?(x):(x%m)) /**< Simple wrap */
#define sWRAP(x,m) (((x)<m)?(((x)<0)?(m+(x)):(x)):(m-(x))) /**<signed integer wrap (no modulus) for negatives where 2m > |x| > m */

//PADDING WILL ONLY AVOID SM CONFLICTS FOR 32BIT
//SM_OFFSET REQUIRED AS FERMI STARTS INDEXING MEMORY FROM LOCATION 0 (i.e. NULL)??
__constant__ int d_SM_START;
__constant__ int d_PADDING;

//SM addressing macro to avoid conflicts (32 bit only)
#define SHARE_INDEX(i, s) ((((s) + d_PADDING)* (i))+d_SM_START) /**<offset struct size by padding to avoid bank conflicts */

//if doubel support is needed then define the following function which requires sm_13 or later
#ifdef _DOUBLE_SUPPORT_REQUIRED_
__inline__ __device__ double tex1DfetchDouble(texture<int2, 1, hipReadModeElementType> tex, int i)
{
	int2 v = tex1Dfetch(tex, i);
  //IF YOU HAVE AN ERROR HERE THEN YOU ARE USING DOUBLE VALUES IN AGENT MEMORY AND NOT COMPILING FOR DOUBLE SUPPORTED HARDWARE
  //To compile for double supported hardware change the CUDA Build rule property "Use sm_13 Architecture (double support)" on the CUDA-Specific Propert Page of the CUDA Build Rule for simulation.cu
	return __hiloint2double(v.y, v.x);
}
#endif

/* Helper functions */
/** next_cell
 * Function used for finding the next cell when using spatial partitioning
 * Upddates the relative cell variable which can have value of -1, 0 or +1
 * @param relative_cell pointer to the relative cell position
 * @return boolean if there is a next cell. True unless relative_Cell value was 1,1,1
 */
__device__ bool next_cell3D(glm::ivec3* relative_cell)
{
	if (relative_cell->x < 1)
	{
		relative_cell->x++;
		return true;
	}
	relative_cell->x = -1;

	if (relative_cell->y < 1)
	{
		relative_cell->y++;
		return true;
	}
	relative_cell->y = -1;
	
	if (relative_cell->z < 1)
	{
		relative_cell->z++;
		return true;
	}
	relative_cell->z = -1;
	
	return false;
}

/** next_cell2D
 * Function used for finding the next cell when using spatial partitioning. Z component is ignored
 * Upddates the relative cell variable which can have value of -1, 0 or +1
 * @param relative_cell pointer to the relative cell position
 * @return boolean if there is a next cell. True unless relative_Cell value was 1,1
 */
__device__ bool next_cell2D(glm::ivec3* relative_cell)
{
	if (relative_cell->x < 1)
	{
		relative_cell->x++;
		return true;
	}
	relative_cell->x = -1;

	if (relative_cell->y < 1)
	{
		relative_cell->y++;
		return true;
	}
	relative_cell->y = -1;
	
	return false;
}


////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created agent agent functions */

/** reset_agent_scan_input
 * agent agent reset scan input function
 * @param agents The xmachine_memory_agent_list agent list
 */
__global__ void reset_agent_scan_input(xmachine_memory_agent_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_agent_Agents
 * agent scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_agent_list agent list destination
 * @param agents_src xmachine_memory_agent_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_agent_Agents(xmachine_memory_agent_list* agents_dst, xmachine_memory_agent_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->id[output_index] = agents_src->id[index];        
		agents_dst->x[output_index] = agents_src->x[index];        
		agents_dst->y[output_index] = agents_src->y[index];        
		agents_dst->velx[output_index] = agents_src->velx[index];        
		agents_dst->vely[output_index] = agents_src->vely[index];        
		agents_dst->steer_x[output_index] = agents_src->steer_x[index];        
		agents_dst->steer_y[output_index] = agents_src->steer_y[index];        
		agents_dst->height[output_index] = agents_src->height[index];        
		agents_dst->exit_no[output_index] = agents_src->exit_no[index];        
		agents_dst->speed[output_index] = agents_src->speed[index];        
		agents_dst->lod[output_index] = agents_src->lod[index];        
		agents_dst->animate[output_index] = agents_src->animate[index];        
		agents_dst->animate_dir[output_index] = agents_src->animate_dir[index];        
		agents_dst->estado[output_index] = agents_src->estado[index];        
		agents_dst->tick[output_index] = agents_src->tick[index];        
		agents_dst->estado_movimiento[output_index] = agents_src->estado_movimiento[index];
	}
}

/** append_agent_Agents
 * agent scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_agent_list agent list destination
 * @param agents_src xmachine_memory_agent_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_agent_Agents(xmachine_memory_agent_list* agents_dst, xmachine_memory_agent_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->id[output_index] = agents_src->id[index];
	    agents_dst->x[output_index] = agents_src->x[index];
	    agents_dst->y[output_index] = agents_src->y[index];
	    agents_dst->velx[output_index] = agents_src->velx[index];
	    agents_dst->vely[output_index] = agents_src->vely[index];
	    agents_dst->steer_x[output_index] = agents_src->steer_x[index];
	    agents_dst->steer_y[output_index] = agents_src->steer_y[index];
	    agents_dst->height[output_index] = agents_src->height[index];
	    agents_dst->exit_no[output_index] = agents_src->exit_no[index];
	    agents_dst->speed[output_index] = agents_src->speed[index];
	    agents_dst->lod[output_index] = agents_src->lod[index];
	    agents_dst->animate[output_index] = agents_src->animate[index];
	    agents_dst->animate_dir[output_index] = agents_src->animate_dir[index];
	    agents_dst->estado[output_index] = agents_src->estado[index];
	    agents_dst->tick[output_index] = agents_src->tick[index];
	    agents_dst->estado_movimiento[output_index] = agents_src->estado_movimiento[index];
    }
}

/** add_agent_agent
 * Continuous agent agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_agent_list to add agents to 
 * @param id agent variable of type unsigned int
 * @param x agent variable of type float
 * @param y agent variable of type float
 * @param velx agent variable of type float
 * @param vely agent variable of type float
 * @param steer_x agent variable of type float
 * @param steer_y agent variable of type float
 * @param height agent variable of type float
 * @param exit_no agent variable of type int
 * @param speed agent variable of type float
 * @param lod agent variable of type int
 * @param animate agent variable of type float
 * @param animate_dir agent variable of type int
 * @param estado agent variable of type int
 * @param tick agent variable of type int
 * @param estado_movimiento agent variable of type unsigned int
 */
template <int AGENT_TYPE>
__device__ void add_agent_agent(xmachine_memory_agent_list* agents, unsigned int id, float x, float y, float velx, float vely, float steer_x, float steer_y, float height, int exit_no, float speed, int lod, float animate, int animate_dir, int estado, int tick, unsigned int estado_movimiento){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->id[index] = id;
	agents->x[index] = x;
	agents->y[index] = y;
	agents->velx[index] = velx;
	agents->vely[index] = vely;
	agents->steer_x[index] = steer_x;
	agents->steer_y[index] = steer_y;
	agents->height[index] = height;
	agents->exit_no[index] = exit_no;
	agents->speed[index] = speed;
	agents->lod[index] = lod;
	agents->animate[index] = animate;
	agents->animate_dir[index] = animate_dir;
	agents->estado[index] = estado;
	agents->tick[index] = tick;
	agents->estado_movimiento[index] = estado_movimiento;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_agent_agent(xmachine_memory_agent_list* agents, unsigned int id, float x, float y, float velx, float vely, float steer_x, float steer_y, float height, int exit_no, float speed, int lod, float animate, int animate_dir, int estado, int tick, unsigned int estado_movimiento){
    add_agent_agent<DISCRETE_2D>(agents, id, x, y, velx, vely, steer_x, steer_y, height, exit_no, speed, lod, animate, animate_dir, estado, tick, estado_movimiento);
}

/** reorder_agent_agents
 * Continuous agent agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_agent_agents(unsigned int* values, xmachine_memory_agent_list* unordered_agents, xmachine_memory_agent_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->id[index] = unordered_agents->id[old_pos];
	ordered_agents->x[index] = unordered_agents->x[old_pos];
	ordered_agents->y[index] = unordered_agents->y[old_pos];
	ordered_agents->velx[index] = unordered_agents->velx[old_pos];
	ordered_agents->vely[index] = unordered_agents->vely[old_pos];
	ordered_agents->steer_x[index] = unordered_agents->steer_x[old_pos];
	ordered_agents->steer_y[index] = unordered_agents->steer_y[old_pos];
	ordered_agents->height[index] = unordered_agents->height[old_pos];
	ordered_agents->exit_no[index] = unordered_agents->exit_no[old_pos];
	ordered_agents->speed[index] = unordered_agents->speed[old_pos];
	ordered_agents->lod[index] = unordered_agents->lod[old_pos];
	ordered_agents->animate[index] = unordered_agents->animate[old_pos];
	ordered_agents->animate_dir[index] = unordered_agents->animate_dir[old_pos];
	ordered_agents->estado[index] = unordered_agents->estado[old_pos];
	ordered_agents->tick[index] = unordered_agents->tick[old_pos];
	ordered_agents->estado_movimiento[index] = unordered_agents->estado_movimiento[old_pos];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created medic agent functions */

/** reset_medic_scan_input
 * medic agent reset scan input function
 * @param agents The xmachine_memory_medic_list agent list
 */
__global__ void reset_medic_scan_input(xmachine_memory_medic_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_medic_Agents
 * medic scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_medic_list agent list destination
 * @param agents_src xmachine_memory_medic_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_medic_Agents(xmachine_memory_medic_list* agents_dst, xmachine_memory_medic_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->x[output_index] = agents_src->x[index];        
		agents_dst->y[output_index] = agents_src->y[index];
	}
}

/** append_medic_Agents
 * medic scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_medic_list agent list destination
 * @param agents_src xmachine_memory_medic_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_medic_Agents(xmachine_memory_medic_list* agents_dst, xmachine_memory_medic_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->x[output_index] = agents_src->x[index];
	    agents_dst->y[output_index] = agents_src->y[index];
    }
}

/** add_medic_agent
 * Continuous medic agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_medic_list to add agents to 
 * @param x agent variable of type int
 * @param y agent variable of type int
 */
template <int AGENT_TYPE>
__device__ void add_medic_agent(xmachine_memory_medic_list* agents, int x, int y){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->x[index] = x;
	agents->y[index] = y;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_medic_agent(xmachine_memory_medic_list* agents, int x, int y){
    add_medic_agent<DISCRETE_2D>(agents, x, y);
}

/** reorder_medic_agents
 * Continuous medic agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_medic_agents(unsigned int* values, xmachine_memory_medic_list* unordered_agents, xmachine_memory_medic_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->x[index] = unordered_agents->x[old_pos];
	ordered_agents->y[index] = unordered_agents->y[old_pos];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created receptionist agent functions */

/** reset_receptionist_scan_input
 * receptionist agent reset scan input function
 * @param agents The xmachine_memory_receptionist_list agent list
 */
__global__ void reset_receptionist_scan_input(xmachine_memory_receptionist_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_receptionist_Agents
 * receptionist scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_receptionist_list agent list destination
 * @param agents_src xmachine_memory_receptionist_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_receptionist_Agents(xmachine_memory_receptionist_list* agents_dst, xmachine_memory_receptionist_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->x[output_index] = agents_src->x[index];        
		agents_dst->y[output_index] = agents_src->y[index];
	    for (int i=0; i<2000; i++){
	      agents_dst->colaPacientes[(i*xmachine_memory_receptionist_MAX)+output_index] = agents_src->colaPacientes[(i*xmachine_memory_receptionist_MAX)+index];
	    }        
		agents_dst->front[output_index] = agents_src->front[index];        
		agents_dst->rear[output_index] = agents_src->rear[index];        
		agents_dst->size[output_index] = agents_src->size[index];
	}
}

/** append_receptionist_Agents
 * receptionist scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_receptionist_list agent list destination
 * @param agents_src xmachine_memory_receptionist_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_receptionist_Agents(xmachine_memory_receptionist_list* agents_dst, xmachine_memory_receptionist_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->x[output_index] = agents_src->x[index];
	    agents_dst->y[output_index] = agents_src->y[index];
	    for (int i=0; i<2000; i++){
	      agents_dst->colaPacientes[(i*xmachine_memory_receptionist_MAX)+output_index] = agents_src->colaPacientes[(i*xmachine_memory_receptionist_MAX)+index];
	    }
	    agents_dst->front[output_index] = agents_src->front[index];
	    agents_dst->rear[output_index] = agents_src->rear[index];
	    agents_dst->size[output_index] = agents_src->size[index];
    }
}

/** add_receptionist_agent
 * Continuous receptionist agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_receptionist_list to add agents to 
 * @param x agent variable of type int
 * @param y agent variable of type int
 * @param colaPacientes agent variable of type unsigned int
 * @param front agent variable of type unsigned int
 * @param rear agent variable of type unsigned int
 * @param size agent variable of type unsigned int
 */
template <int AGENT_TYPE>
__device__ void add_receptionist_agent(xmachine_memory_receptionist_list* agents, int x, int y, unsigned int front, unsigned int rear, unsigned int size){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->x[index] = x;
	agents->y[index] = y;
	agents->front[index] = front;
	agents->rear[index] = rear;
	agents->size[index] = size;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_receptionist_agent(xmachine_memory_receptionist_list* agents, int x, int y, unsigned int front, unsigned int rear, unsigned int size){
    add_receptionist_agent<DISCRETE_2D>(agents, x, y, front, rear, size);
}

/** reorder_receptionist_agents
 * Continuous receptionist agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_receptionist_agents(unsigned int* values, xmachine_memory_receptionist_list* unordered_agents, xmachine_memory_receptionist_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->x[index] = unordered_agents->x[old_pos];
	ordered_agents->y[index] = unordered_agents->y[old_pos];
	for (int i=0; i<2000; i++){
	  ordered_agents->colaPacientes[(i*xmachine_memory_receptionist_MAX)+index] = unordered_agents->colaPacientes[(i*xmachine_memory_receptionist_MAX)+old_pos];
	}
	ordered_agents->front[index] = unordered_agents->front[old_pos];
	ordered_agents->rear[index] = unordered_agents->rear[old_pos];
	ordered_agents->size[index] = unordered_agents->size[old_pos];
}

/** get_receptionist_agent_array_value
 *  Template function for accessing receptionist agent array memory variables. Assumes array points to the first element of the agents array values (offset by agent index)
 *  @param array Agent memory array
 *  @param index to lookup
 *  @return return value
 */
template<typename T>
__FLAME_GPU_FUNC__ T get_receptionist_agent_array_value(T *array, uint index){
	// Null check for out of bounds agents (brute force communication. )
	if(array != nullptr){
	    return array[index*xmachine_memory_receptionist_MAX];
    } else {
    	// Return the default value for this data type 
	    return 0;
    }
}

/** set_receptionist_agent_array_value
 *  Template function for setting receptionist agent array memory variables. Assumes array points to the first element of the agents array values (offset by agent index)
 *  @param array Agent memory array
 *  @param index to lookup
 *  @param return value
 */
template<typename T>
__FLAME_GPU_FUNC__ void set_receptionist_agent_array_value(T *array, uint index, T value){
	// Null check for out of bounds agents (brute force communication. )
	if(array != nullptr){
	    array[index*xmachine_memory_receptionist_MAX] = value;
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created navmap agent functions */

/** reset_navmap_scan_input
 * navmap agent reset scan input function
 * @param agents The xmachine_memory_navmap_list agent list
 */
__global__ void reset_navmap_scan_input(xmachine_memory_navmap_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created pedestrian_location message functions */


/** add_pedestrian_location_message
 * Add non partitioned or spatially partitioned pedestrian_location message
 * @param messages xmachine_message_pedestrian_location_list message list to add too
 * @param x agent variable of type float
 * @param y agent variable of type float
 * @param z agent variable of type float
 * @param estado agent variable of type int
 */
__device__ void add_pedestrian_location_message(xmachine_message_pedestrian_location_list* messages, float x, float y, float z, int estado){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x + d_message_pedestrian_location_count;

	int _position;
	int _scan_input;

	//decide output position
	if(d_message_pedestrian_location_output_type == single_message){
		_position = index; //same as agent position
		_scan_input = 0;
	}else if (d_message_pedestrian_location_output_type == optional_message){
		_position = 0;	   //to be calculated using Prefix sum
		_scan_input = 1;
	}

	//AoS - xmachine_message_pedestrian_location Coalesced memory write
	messages->_scan_input[index] = _scan_input;	
	messages->_position[index] = _position;
	messages->x[index] = x;
	messages->y[index] = y;
	messages->z[index] = z;
	messages->estado[index] = estado;

}

/**
 * Scatter non partitioned or spatially partitioned pedestrian_location message (for optional messages)
 * @param messages scatter_optional_pedestrian_location_messages Sparse xmachine_message_pedestrian_location_list message list
 * @param message_swap temp xmachine_message_pedestrian_location_list message list to scatter sparse messages to
 */
__global__ void scatter_optional_pedestrian_location_messages(xmachine_message_pedestrian_location_list* messages, xmachine_message_pedestrian_location_list* messages_swap){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = messages_swap->_scan_input[index];

	//if optional message is to be written
	if (_scan_input == 1){
		int output_index = messages_swap->_position[index] + d_message_pedestrian_location_count;

		//AoS - xmachine_message_pedestrian_location Un-Coalesced scattered memory write
		messages->_position[output_index] = output_index;
		messages->x[output_index] = messages_swap->x[index];
		messages->y[output_index] = messages_swap->y[index];
		messages->z[output_index] = messages_swap->z[index];
		messages->estado[output_index] = messages_swap->estado[index];				
	}
}

/** reset_pedestrian_location_swaps
 * Reset non partitioned or spatially partitioned pedestrian_location message swaps (for scattering optional messages)
 * @param message_swap message list to reset _position and _scan_input values back to 0
 */
__global__ void reset_pedestrian_location_swaps(xmachine_message_pedestrian_location_list* messages_swap){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	messages_swap->_position[index] = 0;
	messages_swap->_scan_input[index] = 0;
}

/* Message functions */

/** message_pedestrian_location_grid_position
 * Calculates the grid cell position given an glm::vec3 vector
 * @param position glm::vec3 vector representing a position
 */
__device__ glm::ivec3 message_pedestrian_location_grid_position(glm::vec3 position)
{
    glm::ivec3 gridPos;
    gridPos.x = floor((position.x - d_message_pedestrian_location_min_bounds.x) * (float)d_message_pedestrian_location_partitionDim.x / (d_message_pedestrian_location_max_bounds.x - d_message_pedestrian_location_min_bounds.x));
    gridPos.y = floor((position.y - d_message_pedestrian_location_min_bounds.y) * (float)d_message_pedestrian_location_partitionDim.y / (d_message_pedestrian_location_max_bounds.y - d_message_pedestrian_location_min_bounds.y));
    gridPos.z = floor((position.z - d_message_pedestrian_location_min_bounds.z) * (float)d_message_pedestrian_location_partitionDim.z / (d_message_pedestrian_location_max_bounds.z - d_message_pedestrian_location_min_bounds.z));

	//do wrapping or bounding
	

    return gridPos;
}

/** message_pedestrian_location_hash
 * Given the grid position in partition space this function calculates a hash value
 * @param gridPos The position in partition space
 */
__device__ unsigned int message_pedestrian_location_hash(glm::ivec3 gridPos)
{
	//cheap bounding without mod (within range +- partition dimension)
	gridPos.x = (gridPos.x<0)? d_message_pedestrian_location_partitionDim.x-1: gridPos.x; 
	gridPos.x = (gridPos.x>=d_message_pedestrian_location_partitionDim.x)? 0 : gridPos.x; 
	gridPos.y = (gridPos.y<0)? d_message_pedestrian_location_partitionDim.y-1 : gridPos.y; 
	gridPos.y = (gridPos.y>=d_message_pedestrian_location_partitionDim.y)? 0 : gridPos.y; 
	gridPos.z = (gridPos.z<0)? d_message_pedestrian_location_partitionDim.z-1: gridPos.z; 
	gridPos.z = (gridPos.z>=d_message_pedestrian_location_partitionDim.z)? 0 : gridPos.z; 

	//unique id
	return ((gridPos.z * d_message_pedestrian_location_partitionDim.y) * d_message_pedestrian_location_partitionDim.x) + (gridPos.y * d_message_pedestrian_location_partitionDim.x) + gridPos.x;
}

#ifdef FAST_ATOMIC_SORTING
	/** hist_pedestrian_location_messages
		 * Kernal function for performing a histogram (count) on each partition bin and saving the hash and index of a message within that bin
		 * @param local_bin_index output index of the message within the calculated bin
		 * @param unsorted_index output bin index (hash) value
		 * @param messages the message list used to generate the hash value outputs
		 * @param agent_count the current number of agents outputting messages
		 */
	__global__ void hist_pedestrian_location_messages(uint* local_bin_index, uint* unsorted_index, int* global_bin_count, xmachine_message_pedestrian_location_list* messages, int agent_count)
	{
		unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;

		if (index >= agent_count)
			return;
        glm::vec3 position = glm::vec3(messages->x[index], messages->y[index], messages->z[index]);
		glm::ivec3 grid_position = message_pedestrian_location_grid_position(position);
		unsigned int hash = message_pedestrian_location_hash(grid_position);
		unsigned int bin_idx = atomicInc((unsigned int*) &global_bin_count[hash], 0xFFFFFFFF);
		local_bin_index[index] = bin_idx;
		unsorted_index[index] = hash;
	}
	
	/** reorder_pedestrian_location_messages
	 * Reorders the messages accoring to the partition boundary matrix start indices of each bin
	 * @param local_bin_index index of the message within the desired bin
	 * @param unsorted_index bin index (hash) value
	 * @param pbm_start_index the start indices of the partition boundary matrix
	 * @param unordered_messages the original unordered message data
	 * @param ordered_messages buffer used to scatter messages into the correct order
	  @param agent_count the current number of agents outputting messages
	 */
	 __global__ void reorder_pedestrian_location_messages(uint* local_bin_index, uint* unsorted_index, int* pbm_start_index, xmachine_message_pedestrian_location_list* unordered_messages, xmachine_message_pedestrian_location_list* ordered_messages, int agent_count)
	{
		int index = (blockIdx.x *blockDim.x) + threadIdx.x;

		if (index >= agent_count)
			return;

		uint i = unsorted_index[index];
		unsigned int sorted_index = local_bin_index[index] + pbm_start_index[i];

		//finally reorder agent data
		ordered_messages->x[sorted_index] = unordered_messages->x[index];
		ordered_messages->y[sorted_index] = unordered_messages->y[index];
		ordered_messages->z[sorted_index] = unordered_messages->z[index];
		ordered_messages->estado[sorted_index] = unordered_messages->estado[index];
	}
	 
#else

	/** hash_pedestrian_location_messages
	 * Kernal function for calculating a hash value for each messahe depending on its position
	 * @param keys output for the hash key
	 * @param values output for the index value
	 * @param messages the message list used to generate the hash value outputs
	 */
	__global__ void hash_pedestrian_location_messages(uint* keys, uint* values, xmachine_message_pedestrian_location_list* messages)
	{
		unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
        glm::vec3 position = glm::vec3(messages->x[index], messages->y[index], messages->z[index]);
		glm::ivec3 grid_position = message_pedestrian_location_grid_position(position);
		unsigned int hash = message_pedestrian_location_hash(grid_position);

		keys[index] = hash;
		values[index] = index;
	}

	/** reorder_pedestrian_location_messages
	 * Reorders the messages accoring to the ordered sort identifiers and builds a Partition Boundary Matrix by looking at the previosu threads sort id.
	 * @param keys the sorted hash keys
	 * @param values the sorted index values
	 * @param matrix the PBM
	 * @param unordered_messages the original unordered message data
	 * @param ordered_messages buffer used to scatter messages into the correct order
	 */
	__global__ void reorder_pedestrian_location_messages(uint* keys, uint* values, xmachine_message_pedestrian_location_PBM* matrix, xmachine_message_pedestrian_location_list* unordered_messages, xmachine_message_pedestrian_location_list* ordered_messages)
	{
		extern __shared__ int sm_data [];

		int index = (blockIdx.x * blockDim.x) + threadIdx.x;

		//load threads sort key into sm
		uint key = keys[index];
		uint old_pos = values[index];

		sm_data[threadIdx.x] = key;
		__syncthreads();
	
		unsigned int prev_key;

		//if first thread then no prev sm value so get prev from global memory 
		if (threadIdx.x == 0)
		{
			//first thread has no prev value so ignore
			if (index != 0)
				prev_key = keys[index-1];
		}
		//get previous ident from sm
		else	
		{
			prev_key = sm_data[threadIdx.x-1];
		}

		//TODO: Check key is not out of bounds

		//set partition boundaries
		if (index < d_message_pedestrian_location_count)
		{
			//if first thread then set first partition cell start
			if (index == 0)
			{
				matrix->start[key] = index;
			}

			//if edge of a boundr update start and end of partition
			else if (prev_key != key)
			{
				//set start for key
				matrix->start[key] = index;

				//set end for key -1
				matrix->end_or_count[prev_key] = index;
			}

			//if last thread then set final partition cell end
			if (index == d_message_pedestrian_location_count-1)
			{
				matrix->end_or_count[key] = index+1;
			}
		}
	
		//finally reorder agent data
		ordered_messages->x[index] = unordered_messages->x[old_pos];
		ordered_messages->y[index] = unordered_messages->y[old_pos];
		ordered_messages->z[index] = unordered_messages->z[old_pos];
		ordered_messages->estado[index] = unordered_messages->estado[old_pos];
	}

#endif

/** load_next_pedestrian_location_message
 * Used to load the next message data to shared memory
 * Idea is check the current cell index to see if we can simply get a message from the current cell
 * If we are at the end of the current cell then loop till we find the next cell with messages (this way we ignore cells with no messages)
 * @param messages the message list
 * @param partition_matrix the PBM
 * @param relative_cell the relative partition cell position from the agent position
 * @param cell_index_max the maximum index of the current partition cell
 * @param agent_grid_cell the agents partition cell position
 * @param cell_index the current cell index in agent_grid_cell+relative_cell
 * @return true if a message has been loaded into sm false otherwise
 */
__device__ bool load_next_pedestrian_location_message(xmachine_message_pedestrian_location_list* messages, xmachine_message_pedestrian_location_PBM* partition_matrix, glm::ivec3 relative_cell, int cell_index_max, glm::ivec3 agent_grid_cell, int cell_index)
{
	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];

	int move_cell = true;
	cell_index ++;

	//see if we need to move to a new partition cell
	if(cell_index < cell_index_max)
		move_cell = false;

	while(move_cell)
	{
		//get the next relative grid position 
        if (next_cell2D(&relative_cell))
		{
			//calculate the next cells grid position and hash
			glm::ivec3 next_cell_position = agent_grid_cell + relative_cell;
			int next_cell_hash = message_pedestrian_location_hash(next_cell_position);
			//use the hash to calculate the start index
			int cell_index_min = tex1Dfetch(tex_xmachine_message_pedestrian_location_pbm_start, next_cell_hash + d_tex_xmachine_message_pedestrian_location_pbm_start_offset);
			cell_index_max = tex1Dfetch(tex_xmachine_message_pedestrian_location_pbm_end_or_count, next_cell_hash + d_tex_xmachine_message_pedestrian_location_pbm_end_or_count_offset);
			//check for messages in the cell (cell index max is the count for atomic sorting)
#ifdef FAST_ATOMIC_SORTING
			if (cell_index_max > 0)
			{
				//when using fast atomics value represents bin count not last index!
				cell_index_max += cell_index_min; //when using fast atomics value represents bin count not last index!
#else
			if (cell_index_min != 0xffffffff)
			{
#endif
				//start from the cell index min
				cell_index = cell_index_min;
				//exit the loop as we have found a valid cell with message data
				move_cell = false;
			}
		}
		else
		{
			//we have exhausted all the neighbouring cells so there are no more messages
			return false;
		}
	}
	
	//get message data using texture fetch
	xmachine_message_pedestrian_location temp_message;
	temp_message._relative_cell = relative_cell;
	temp_message._cell_index_max = cell_index_max;
	temp_message._cell_index = cell_index;
	temp_message._agent_grid_cell = agent_grid_cell;

	//Using texture cache
  temp_message.x = tex1Dfetch(tex_xmachine_message_pedestrian_location_x, cell_index + d_tex_xmachine_message_pedestrian_location_x_offset); temp_message.y = tex1Dfetch(tex_xmachine_message_pedestrian_location_y, cell_index + d_tex_xmachine_message_pedestrian_location_y_offset); temp_message.z = tex1Dfetch(tex_xmachine_message_pedestrian_location_z, cell_index + d_tex_xmachine_message_pedestrian_location_z_offset); temp_message.estado = tex1Dfetch(tex_xmachine_message_pedestrian_location_estado, cell_index + d_tex_xmachine_message_pedestrian_location_estado_offset); 

	//load it into shared memory (no sync as no sharing between threads)
	int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_pedestrian_location));
	xmachine_message_pedestrian_location* sm_message = ((xmachine_message_pedestrian_location*)&message_share[message_index]);
	sm_message[0] = temp_message;

	return true;
}


/*
 * get first spatial partitioned pedestrian_location message (first batch load into shared memory)
 */
__device__ xmachine_message_pedestrian_location* get_first_pedestrian_location_message(xmachine_message_pedestrian_location_list* messages, xmachine_message_pedestrian_location_PBM* partition_matrix, float x, float y, float z){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];

	// If there are no messages, do not load any messages
	if(d_message_pedestrian_location_count == 0){
		return nullptr;
	}

	glm::ivec3 relative_cell = glm::ivec3(-2, -1, -1);
	int cell_index_max = 0;
	int cell_index = 0;
	glm::vec3 position = glm::vec3(x, y, z);
	glm::ivec3 agent_grid_cell = message_pedestrian_location_grid_position(position);
	
	if (load_next_pedestrian_location_message(messages, partition_matrix, relative_cell, cell_index_max, agent_grid_cell, cell_index))
	{
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_pedestrian_location));
		return ((xmachine_message_pedestrian_location*)&message_share[message_index]);
	}
	else
	{
		return nullptr;
	}
}

/*
 * get next spatial partitioned pedestrian_location message (either from SM or next batch load)
 */
__device__ xmachine_message_pedestrian_location* get_next_pedestrian_location_message(xmachine_message_pedestrian_location* message, xmachine_message_pedestrian_location_list* messages, xmachine_message_pedestrian_location_PBM* partition_matrix){
	
	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	// If there are no messages, do not load any messages
	if(d_message_pedestrian_location_count == 0){
		return nullptr;
	}
	
	if (load_next_pedestrian_location_message(messages, partition_matrix, message->_relative_cell, message->_cell_index_max, message->_agent_grid_cell, message->_cell_index))
	{
		//get conflict free address of 
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_pedestrian_location));
		return ((xmachine_message_pedestrian_location*)&message_share[message_index]);
	}
	else
		return nullptr;
	
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created pedestrian_state message functions */


/** add_pedestrian_state_message
 * Add non partitioned or spatially partitioned pedestrian_state message
 * @param messages xmachine_message_pedestrian_state_list message list to add too
 * @param x agent variable of type float
 * @param y agent variable of type float
 * @param z agent variable of type float
 * @param estado agent variable of type int
 */
__device__ void add_pedestrian_state_message(xmachine_message_pedestrian_state_list* messages, float x, float y, float z, int estado){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x + d_message_pedestrian_state_count;

	int _position;
	int _scan_input;

	//decide output position
	if(d_message_pedestrian_state_output_type == single_message){
		_position = index; //same as agent position
		_scan_input = 0;
	}else if (d_message_pedestrian_state_output_type == optional_message){
		_position = 0;	   //to be calculated using Prefix sum
		_scan_input = 1;
	}

	//AoS - xmachine_message_pedestrian_state Coalesced memory write
	messages->_scan_input[index] = _scan_input;	
	messages->_position[index] = _position;
	messages->x[index] = x;
	messages->y[index] = y;
	messages->z[index] = z;
	messages->estado[index] = estado;

}

/**
 * Scatter non partitioned or spatially partitioned pedestrian_state message (for optional messages)
 * @param messages scatter_optional_pedestrian_state_messages Sparse xmachine_message_pedestrian_state_list message list
 * @param message_swap temp xmachine_message_pedestrian_state_list message list to scatter sparse messages to
 */
__global__ void scatter_optional_pedestrian_state_messages(xmachine_message_pedestrian_state_list* messages, xmachine_message_pedestrian_state_list* messages_swap){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = messages_swap->_scan_input[index];

	//if optional message is to be written
	if (_scan_input == 1){
		int output_index = messages_swap->_position[index] + d_message_pedestrian_state_count;

		//AoS - xmachine_message_pedestrian_state Un-Coalesced scattered memory write
		messages->_position[output_index] = output_index;
		messages->x[output_index] = messages_swap->x[index];
		messages->y[output_index] = messages_swap->y[index];
		messages->z[output_index] = messages_swap->z[index];
		messages->estado[output_index] = messages_swap->estado[index];				
	}
}

/** reset_pedestrian_state_swaps
 * Reset non partitioned or spatially partitioned pedestrian_state message swaps (for scattering optional messages)
 * @param message_swap message list to reset _position and _scan_input values back to 0
 */
__global__ void reset_pedestrian_state_swaps(xmachine_message_pedestrian_state_list* messages_swap){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	messages_swap->_position[index] = 0;
	messages_swap->_scan_input[index] = 0;
}

/* Message functions */

/** message_pedestrian_state_grid_position
 * Calculates the grid cell position given an glm::vec3 vector
 * @param position glm::vec3 vector representing a position
 */
__device__ glm::ivec3 message_pedestrian_state_grid_position(glm::vec3 position)
{
    glm::ivec3 gridPos;
    gridPos.x = floor((position.x - d_message_pedestrian_state_min_bounds.x) * (float)d_message_pedestrian_state_partitionDim.x / (d_message_pedestrian_state_max_bounds.x - d_message_pedestrian_state_min_bounds.x));
    gridPos.y = floor((position.y - d_message_pedestrian_state_min_bounds.y) * (float)d_message_pedestrian_state_partitionDim.y / (d_message_pedestrian_state_max_bounds.y - d_message_pedestrian_state_min_bounds.y));
    gridPos.z = floor((position.z - d_message_pedestrian_state_min_bounds.z) * (float)d_message_pedestrian_state_partitionDim.z / (d_message_pedestrian_state_max_bounds.z - d_message_pedestrian_state_min_bounds.z));

	//do wrapping or bounding
	

    return gridPos;
}

/** message_pedestrian_state_hash
 * Given the grid position in partition space this function calculates a hash value
 * @param gridPos The position in partition space
 */
__device__ unsigned int message_pedestrian_state_hash(glm::ivec3 gridPos)
{
	//cheap bounding without mod (within range +- partition dimension)
	gridPos.x = (gridPos.x<0)? d_message_pedestrian_state_partitionDim.x-1: gridPos.x; 
	gridPos.x = (gridPos.x>=d_message_pedestrian_state_partitionDim.x)? 0 : gridPos.x; 
	gridPos.y = (gridPos.y<0)? d_message_pedestrian_state_partitionDim.y-1 : gridPos.y; 
	gridPos.y = (gridPos.y>=d_message_pedestrian_state_partitionDim.y)? 0 : gridPos.y; 
	gridPos.z = (gridPos.z<0)? d_message_pedestrian_state_partitionDim.z-1: gridPos.z; 
	gridPos.z = (gridPos.z>=d_message_pedestrian_state_partitionDim.z)? 0 : gridPos.z; 

	//unique id
	return ((gridPos.z * d_message_pedestrian_state_partitionDim.y) * d_message_pedestrian_state_partitionDim.x) + (gridPos.y * d_message_pedestrian_state_partitionDim.x) + gridPos.x;
}

#ifdef FAST_ATOMIC_SORTING
	/** hist_pedestrian_state_messages
		 * Kernal function for performing a histogram (count) on each partition bin and saving the hash and index of a message within that bin
		 * @param local_bin_index output index of the message within the calculated bin
		 * @param unsorted_index output bin index (hash) value
		 * @param messages the message list used to generate the hash value outputs
		 * @param agent_count the current number of agents outputting messages
		 */
	__global__ void hist_pedestrian_state_messages(uint* local_bin_index, uint* unsorted_index, int* global_bin_count, xmachine_message_pedestrian_state_list* messages, int agent_count)
	{
		unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;

		if (index >= agent_count)
			return;
        glm::vec3 position = glm::vec3(messages->x[index], messages->y[index], messages->z[index]);
		glm::ivec3 grid_position = message_pedestrian_state_grid_position(position);
		unsigned int hash = message_pedestrian_state_hash(grid_position);
		unsigned int bin_idx = atomicInc((unsigned int*) &global_bin_count[hash], 0xFFFFFFFF);
		local_bin_index[index] = bin_idx;
		unsorted_index[index] = hash;
	}
	
	/** reorder_pedestrian_state_messages
	 * Reorders the messages accoring to the partition boundary matrix start indices of each bin
	 * @param local_bin_index index of the message within the desired bin
	 * @param unsorted_index bin index (hash) value
	 * @param pbm_start_index the start indices of the partition boundary matrix
	 * @param unordered_messages the original unordered message data
	 * @param ordered_messages buffer used to scatter messages into the correct order
	  @param agent_count the current number of agents outputting messages
	 */
	 __global__ void reorder_pedestrian_state_messages(uint* local_bin_index, uint* unsorted_index, int* pbm_start_index, xmachine_message_pedestrian_state_list* unordered_messages, xmachine_message_pedestrian_state_list* ordered_messages, int agent_count)
	{
		int index = (blockIdx.x *blockDim.x) + threadIdx.x;

		if (index >= agent_count)
			return;

		uint i = unsorted_index[index];
		unsigned int sorted_index = local_bin_index[index] + pbm_start_index[i];

		//finally reorder agent data
		ordered_messages->x[sorted_index] = unordered_messages->x[index];
		ordered_messages->y[sorted_index] = unordered_messages->y[index];
		ordered_messages->z[sorted_index] = unordered_messages->z[index];
		ordered_messages->estado[sorted_index] = unordered_messages->estado[index];
	}
	 
#else

	/** hash_pedestrian_state_messages
	 * Kernal function for calculating a hash value for each messahe depending on its position
	 * @param keys output for the hash key
	 * @param values output for the index value
	 * @param messages the message list used to generate the hash value outputs
	 */
	__global__ void hash_pedestrian_state_messages(uint* keys, uint* values, xmachine_message_pedestrian_state_list* messages)
	{
		unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
        glm::vec3 position = glm::vec3(messages->x[index], messages->y[index], messages->z[index]);
		glm::ivec3 grid_position = message_pedestrian_state_grid_position(position);
		unsigned int hash = message_pedestrian_state_hash(grid_position);

		keys[index] = hash;
		values[index] = index;
	}

	/** reorder_pedestrian_state_messages
	 * Reorders the messages accoring to the ordered sort identifiers and builds a Partition Boundary Matrix by looking at the previosu threads sort id.
	 * @param keys the sorted hash keys
	 * @param values the sorted index values
	 * @param matrix the PBM
	 * @param unordered_messages the original unordered message data
	 * @param ordered_messages buffer used to scatter messages into the correct order
	 */
	__global__ void reorder_pedestrian_state_messages(uint* keys, uint* values, xmachine_message_pedestrian_state_PBM* matrix, xmachine_message_pedestrian_state_list* unordered_messages, xmachine_message_pedestrian_state_list* ordered_messages)
	{
		extern __shared__ int sm_data [];

		int index = (blockIdx.x * blockDim.x) + threadIdx.x;

		//load threads sort key into sm
		uint key = keys[index];
		uint old_pos = values[index];

		sm_data[threadIdx.x] = key;
		__syncthreads();
	
		unsigned int prev_key;

		//if first thread then no prev sm value so get prev from global memory 
		if (threadIdx.x == 0)
		{
			//first thread has no prev value so ignore
			if (index != 0)
				prev_key = keys[index-1];
		}
		//get previous ident from sm
		else	
		{
			prev_key = sm_data[threadIdx.x-1];
		}

		//TODO: Check key is not out of bounds

		//set partition boundaries
		if (index < d_message_pedestrian_state_count)
		{
			//if first thread then set first partition cell start
			if (index == 0)
			{
				matrix->start[key] = index;
			}

			//if edge of a boundr update start and end of partition
			else if (prev_key != key)
			{
				//set start for key
				matrix->start[key] = index;

				//set end for key -1
				matrix->end_or_count[prev_key] = index;
			}

			//if last thread then set final partition cell end
			if (index == d_message_pedestrian_state_count-1)
			{
				matrix->end_or_count[key] = index+1;
			}
		}
	
		//finally reorder agent data
		ordered_messages->x[index] = unordered_messages->x[old_pos];
		ordered_messages->y[index] = unordered_messages->y[old_pos];
		ordered_messages->z[index] = unordered_messages->z[old_pos];
		ordered_messages->estado[index] = unordered_messages->estado[old_pos];
	}

#endif

/** load_next_pedestrian_state_message
 * Used to load the next message data to shared memory
 * Idea is check the current cell index to see if we can simply get a message from the current cell
 * If we are at the end of the current cell then loop till we find the next cell with messages (this way we ignore cells with no messages)
 * @param messages the message list
 * @param partition_matrix the PBM
 * @param relative_cell the relative partition cell position from the agent position
 * @param cell_index_max the maximum index of the current partition cell
 * @param agent_grid_cell the agents partition cell position
 * @param cell_index the current cell index in agent_grid_cell+relative_cell
 * @return true if a message has been loaded into sm false otherwise
 */
__device__ bool load_next_pedestrian_state_message(xmachine_message_pedestrian_state_list* messages, xmachine_message_pedestrian_state_PBM* partition_matrix, glm::ivec3 relative_cell, int cell_index_max, glm::ivec3 agent_grid_cell, int cell_index)
{
	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];

	int move_cell = true;
	cell_index ++;

	//see if we need to move to a new partition cell
	if(cell_index < cell_index_max)
		move_cell = false;

	while(move_cell)
	{
		//get the next relative grid position 
        if (next_cell2D(&relative_cell))
		{
			//calculate the next cells grid position and hash
			glm::ivec3 next_cell_position = agent_grid_cell + relative_cell;
			int next_cell_hash = message_pedestrian_state_hash(next_cell_position);
			//use the hash to calculate the start index
			int cell_index_min = tex1Dfetch(tex_xmachine_message_pedestrian_state_pbm_start, next_cell_hash + d_tex_xmachine_message_pedestrian_state_pbm_start_offset);
			cell_index_max = tex1Dfetch(tex_xmachine_message_pedestrian_state_pbm_end_or_count, next_cell_hash + d_tex_xmachine_message_pedestrian_state_pbm_end_or_count_offset);
			//check for messages in the cell (cell index max is the count for atomic sorting)
#ifdef FAST_ATOMIC_SORTING
			if (cell_index_max > 0)
			{
				//when using fast atomics value represents bin count not last index!
				cell_index_max += cell_index_min; //when using fast atomics value represents bin count not last index!
#else
			if (cell_index_min != 0xffffffff)
			{
#endif
				//start from the cell index min
				cell_index = cell_index_min;
				//exit the loop as we have found a valid cell with message data
				move_cell = false;
			}
		}
		else
		{
			//we have exhausted all the neighbouring cells so there are no more messages
			return false;
		}
	}
	
	//get message data using texture fetch
	xmachine_message_pedestrian_state temp_message;
	temp_message._relative_cell = relative_cell;
	temp_message._cell_index_max = cell_index_max;
	temp_message._cell_index = cell_index;
	temp_message._agent_grid_cell = agent_grid_cell;

	//Using texture cache
  temp_message.x = tex1Dfetch(tex_xmachine_message_pedestrian_state_x, cell_index + d_tex_xmachine_message_pedestrian_state_x_offset); temp_message.y = tex1Dfetch(tex_xmachine_message_pedestrian_state_y, cell_index + d_tex_xmachine_message_pedestrian_state_y_offset); temp_message.z = tex1Dfetch(tex_xmachine_message_pedestrian_state_z, cell_index + d_tex_xmachine_message_pedestrian_state_z_offset); temp_message.estado = tex1Dfetch(tex_xmachine_message_pedestrian_state_estado, cell_index + d_tex_xmachine_message_pedestrian_state_estado_offset); 

	//load it into shared memory (no sync as no sharing between threads)
	int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_pedestrian_state));
	xmachine_message_pedestrian_state* sm_message = ((xmachine_message_pedestrian_state*)&message_share[message_index]);
	sm_message[0] = temp_message;

	return true;
}


/*
 * get first spatial partitioned pedestrian_state message (first batch load into shared memory)
 */
__device__ xmachine_message_pedestrian_state* get_first_pedestrian_state_message(xmachine_message_pedestrian_state_list* messages, xmachine_message_pedestrian_state_PBM* partition_matrix, float x, float y, float z){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];

	// If there are no messages, do not load any messages
	if(d_message_pedestrian_state_count == 0){
		return nullptr;
	}

	glm::ivec3 relative_cell = glm::ivec3(-2, -1, -1);
	int cell_index_max = 0;
	int cell_index = 0;
	glm::vec3 position = glm::vec3(x, y, z);
	glm::ivec3 agent_grid_cell = message_pedestrian_state_grid_position(position);
	
	if (load_next_pedestrian_state_message(messages, partition_matrix, relative_cell, cell_index_max, agent_grid_cell, cell_index))
	{
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_pedestrian_state));
		return ((xmachine_message_pedestrian_state*)&message_share[message_index]);
	}
	else
	{
		return nullptr;
	}
}

/*
 * get next spatial partitioned pedestrian_state message (either from SM or next batch load)
 */
__device__ xmachine_message_pedestrian_state* get_next_pedestrian_state_message(xmachine_message_pedestrian_state* message, xmachine_message_pedestrian_state_list* messages, xmachine_message_pedestrian_state_PBM* partition_matrix){
	
	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	// If there are no messages, do not load any messages
	if(d_message_pedestrian_state_count == 0){
		return nullptr;
	}
	
	if (load_next_pedestrian_state_message(messages, partition_matrix, message->_relative_cell, message->_cell_index_max, message->_agent_grid_cell, message->_cell_index))
	{
		//get conflict free address of 
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_pedestrian_state));
		return ((xmachine_message_pedestrian_state*)&message_share[message_index]);
	}
	else
		return nullptr;
	
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created check_in message functions */


/** add_check_in_message
 * Add non partitioned or spatially partitioned check_in message
 * @param messages xmachine_message_check_in_list message list to add too
 * @param id agent variable of type unsigned int
 */
__device__ void add_check_in_message(xmachine_message_check_in_list* messages, unsigned int id){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x + d_message_check_in_count;

	int _position;
	int _scan_input;

	//decide output position
	if(d_message_check_in_output_type == single_message){
		_position = index; //same as agent position
		_scan_input = 0;
	}else if (d_message_check_in_output_type == optional_message){
		_position = 0;	   //to be calculated using Prefix sum
		_scan_input = 1;
	}

	//AoS - xmachine_message_check_in Coalesced memory write
	messages->_scan_input[index] = _scan_input;	
	messages->_position[index] = _position;
	messages->id[index] = id;

}

/**
 * Scatter non partitioned or spatially partitioned check_in message (for optional messages)
 * @param messages scatter_optional_check_in_messages Sparse xmachine_message_check_in_list message list
 * @param message_swap temp xmachine_message_check_in_list message list to scatter sparse messages to
 */
__global__ void scatter_optional_check_in_messages(xmachine_message_check_in_list* messages, xmachine_message_check_in_list* messages_swap){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = messages_swap->_scan_input[index];

	//if optional message is to be written
	if (_scan_input == 1){
		int output_index = messages_swap->_position[index] + d_message_check_in_count;

		//AoS - xmachine_message_check_in Un-Coalesced scattered memory write
		messages->_position[output_index] = output_index;
		messages->id[output_index] = messages_swap->id[index];				
	}
}

/** reset_check_in_swaps
 * Reset non partitioned or spatially partitioned check_in message swaps (for scattering optional messages)
 * @param message_swap message list to reset _position and _scan_input values back to 0
 */
__global__ void reset_check_in_swaps(xmachine_message_check_in_list* messages_swap){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	messages_swap->_position[index] = 0;
	messages_swap->_scan_input[index] = 0;
}

/* Message functions */

__device__ xmachine_message_check_in* get_first_check_in_message(xmachine_message_check_in_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = (ceil((float)d_message_check_in_count/ blockDim.x)* blockDim.x);

	//if no messages then return a null pointer (false)
	if (wrap_size == 0)
		return nullptr;

	//global thread index
	int global_index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//global thread index
	int index = WRAP(global_index, wrap_size);

	//SoA to AoS - xmachine_message_check_in Coalesced memory read
	xmachine_message_check_in temp_message;
	temp_message._position = messages->_position[index];
	temp_message.id = messages->id[index];

	//AoS to shared memory
	int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_check_in));
	xmachine_message_check_in* sm_message = ((xmachine_message_check_in*)&message_share[message_index]);
	sm_message[0] = temp_message;

	__syncthreads();

  //HACK FOR 64 bit addressing issue in sm
	return ((xmachine_message_check_in*)&message_share[d_SM_START]);
}

__device__ xmachine_message_check_in* get_next_check_in_message(xmachine_message_check_in* message, xmachine_message_check_in_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = ceil((float)d_message_check_in_count/ blockDim.x)*blockDim.x;

	int i = WRAP((message->_position + 1),wrap_size);

	//If end of messages (last message not multiple of gridsize) go to 0 index
	if (i >= d_message_check_in_count)
		i = 0;

	//Check if back to start position of first message
	if (i == WRAP((blockDim.x* blockIdx.x), wrap_size))
		return nullptr;

	int tile = floor((float)i/(blockDim.x)); //tile is round down position over blockDim
	i = i % blockDim.x;						 //mod i for shared memory index

	//if count == Block Size load next tile int shared memory values
	if (i == 0){
		__syncthreads();					//make sure we don't change shared memory until all threads are here (important for emu-debug mode)
		
		//SoA to AoS - xmachine_message_check_in Coalesced memory read
		int index = (tile* blockDim.x) + threadIdx.x;
		xmachine_message_check_in temp_message;
		temp_message._position = messages->_position[index];
		temp_message.id = messages->id[index];

		//AoS to shared memory
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_check_in));
		xmachine_message_check_in* sm_message = ((xmachine_message_check_in*)&message_share[message_index]);
		sm_message[0] = temp_message;

		__syncthreads();					//make sure we don't start returning messages until all threads have updated shared memory
	}

	int message_index = SHARE_INDEX(i, sizeof(xmachine_message_check_in));
	return ((xmachine_message_check_in*)&message_share[message_index]);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created avisar_paciente message functions */


/** add_avisar_paciente_message
 * Add non partitioned or spatially partitioned avisar_paciente message
 * @param messages xmachine_message_avisar_paciente_list message list to add too
 * @param id agent variable of type unsigned int
 */
__device__ void add_avisar_paciente_message(xmachine_message_avisar_paciente_list* messages, unsigned int id){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x + d_message_avisar_paciente_count;

	int _position;
	int _scan_input;

	//decide output position
	if(d_message_avisar_paciente_output_type == single_message){
		_position = index; //same as agent position
		_scan_input = 0;
	}else if (d_message_avisar_paciente_output_type == optional_message){
		_position = 0;	   //to be calculated using Prefix sum
		_scan_input = 1;
	}

	//AoS - xmachine_message_avisar_paciente Coalesced memory write
	messages->_scan_input[index] = _scan_input;	
	messages->_position[index] = _position;
	messages->id[index] = id;

}

/**
 * Scatter non partitioned or spatially partitioned avisar_paciente message (for optional messages)
 * @param messages scatter_optional_avisar_paciente_messages Sparse xmachine_message_avisar_paciente_list message list
 * @param message_swap temp xmachine_message_avisar_paciente_list message list to scatter sparse messages to
 */
__global__ void scatter_optional_avisar_paciente_messages(xmachine_message_avisar_paciente_list* messages, xmachine_message_avisar_paciente_list* messages_swap){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = messages_swap->_scan_input[index];

	//if optional message is to be written
	if (_scan_input == 1){
		int output_index = messages_swap->_position[index] + d_message_avisar_paciente_count;

		//AoS - xmachine_message_avisar_paciente Un-Coalesced scattered memory write
		messages->_position[output_index] = output_index;
		messages->id[output_index] = messages_swap->id[index];				
	}
}

/** reset_avisar_paciente_swaps
 * Reset non partitioned or spatially partitioned avisar_paciente message swaps (for scattering optional messages)
 * @param message_swap message list to reset _position and _scan_input values back to 0
 */
__global__ void reset_avisar_paciente_swaps(xmachine_message_avisar_paciente_list* messages_swap){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	messages_swap->_position[index] = 0;
	messages_swap->_scan_input[index] = 0;
}

/* Message functions */

__device__ xmachine_message_avisar_paciente* get_first_avisar_paciente_message(xmachine_message_avisar_paciente_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = (ceil((float)d_message_avisar_paciente_count/ blockDim.x)* blockDim.x);

	//if no messages then return a null pointer (false)
	if (wrap_size == 0)
		return nullptr;

	//global thread index
	int global_index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//global thread index
	int index = WRAP(global_index, wrap_size);

	//SoA to AoS - xmachine_message_avisar_paciente Coalesced memory read
	xmachine_message_avisar_paciente temp_message;
	temp_message._position = messages->_position[index];
	temp_message.id = messages->id[index];

	//AoS to shared memory
	int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_avisar_paciente));
	xmachine_message_avisar_paciente* sm_message = ((xmachine_message_avisar_paciente*)&message_share[message_index]);
	sm_message[0] = temp_message;

	__syncthreads();

  //HACK FOR 64 bit addressing issue in sm
	return ((xmachine_message_avisar_paciente*)&message_share[d_SM_START]);
}

__device__ xmachine_message_avisar_paciente* get_next_avisar_paciente_message(xmachine_message_avisar_paciente* message, xmachine_message_avisar_paciente_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = ceil((float)d_message_avisar_paciente_count/ blockDim.x)*blockDim.x;

	int i = WRAP((message->_position + 1),wrap_size);

	//If end of messages (last message not multiple of gridsize) go to 0 index
	if (i >= d_message_avisar_paciente_count)
		i = 0;

	//Check if back to start position of first message
	if (i == WRAP((blockDim.x* blockIdx.x), wrap_size))
		return nullptr;

	int tile = floor((float)i/(blockDim.x)); //tile is round down position over blockDim
	i = i % blockDim.x;						 //mod i for shared memory index

	//if count == Block Size load next tile int shared memory values
	if (i == 0){
		__syncthreads();					//make sure we don't change shared memory until all threads are here (important for emu-debug mode)
		
		//SoA to AoS - xmachine_message_avisar_paciente Coalesced memory read
		int index = (tile* blockDim.x) + threadIdx.x;
		xmachine_message_avisar_paciente temp_message;
		temp_message._position = messages->_position[index];
		temp_message.id = messages->id[index];

		//AoS to shared memory
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_avisar_paciente));
		xmachine_message_avisar_paciente* sm_message = ((xmachine_message_avisar_paciente*)&message_share[message_index]);
		sm_message[0] = temp_message;

		__syncthreads();					//make sure we don't start returning messages until all threads have updated shared memory
	}

	int message_index = SHARE_INDEX(i, sizeof(xmachine_message_avisar_paciente));
	return ((xmachine_message_avisar_paciente*)&message_share[message_index]);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created navmap_cell message functions */


/* Message functions */

template <int AGENT_TYPE>
__device__ void add_navmap_cell_message(xmachine_message_navmap_cell_list* messages, int x, int y, int exit_no, float height, float collision_x, float collision_y){
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x * gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x * blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y * blockDim.y) + threadIdx.y;

		int index = global_position.x + (global_position.y * width);

		
		messages->x[index] = x;			
		messages->y[index] = y;			
		messages->exit_no[index] = exit_no;			
		messages->height[index] = height;			
		messages->collision_x[index] = collision_x;			
		messages->collision_y[index] = collision_y;			
	}
	//else CONTINUOUS agents can not write to discrete space
}

//Used by continuous agents this accesses messages with texture cache. agent_x and agent_y are discrete positions in the message space
__device__ xmachine_message_navmap_cell* get_first_navmap_cell_message_continuous(xmachine_message_navmap_cell_list* messages,  int agent_x, int agent_y){

	//shared memory get from offset dependant on sm usage in function
	extern __shared__ int sm_data [];

	xmachine_message_navmap_cell* message_share = (xmachine_message_navmap_cell*)&sm_data[0];
	
	int range = d_message_navmap_cell_range;
	int width = d_message_navmap_cell_width;
	
	glm::ivec2 global_position;
	global_position.x = sWRAP(agent_x-range , width);
	global_position.y = sWRAP(agent_y-range , width);
	

	int index = ((global_position.y)* width) + global_position.x;
	
	xmachine_message_navmap_cell temp_message;
	temp_message._position = glm::ivec2(agent_x, agent_y);
	temp_message._relative = glm::ivec2(-range, -range);

	temp_message.x = tex1Dfetch(tex_xmachine_message_navmap_cell_x, index + d_tex_xmachine_message_navmap_cell_x_offset);temp_message.y = tex1Dfetch(tex_xmachine_message_navmap_cell_y, index + d_tex_xmachine_message_navmap_cell_y_offset);temp_message.exit_no = tex1Dfetch(tex_xmachine_message_navmap_cell_exit_no, index + d_tex_xmachine_message_navmap_cell_exit_no_offset);temp_message.height = tex1Dfetch(tex_xmachine_message_navmap_cell_height, index + d_tex_xmachine_message_navmap_cell_height_offset);temp_message.collision_x = tex1Dfetch(tex_xmachine_message_navmap_cell_collision_x, index + d_tex_xmachine_message_navmap_cell_collision_x_offset);temp_message.collision_y = tex1Dfetch(tex_xmachine_message_navmap_cell_collision_y, index + d_tex_xmachine_message_navmap_cell_collision_y_offset);
	
	message_share[threadIdx.x] = temp_message;

	//return top left of messages
	return &message_share[threadIdx.x];
}

//Get next navmap_cell message  continuous
//Used by continuous agents this accesses messages with texture cache (agent position in discrete space was set when accessing first message)
__device__ xmachine_message_navmap_cell* get_next_navmap_cell_message_continuous(xmachine_message_navmap_cell* message, xmachine_message_navmap_cell_list* messages){

	//shared memory get from offset dependant on sm usage in function
	extern __shared__ int sm_data [];

	xmachine_message_navmap_cell* message_share = (xmachine_message_navmap_cell*)&sm_data[0];
	
	int range = d_message_navmap_cell_range;
	int width = d_message_navmap_cell_width;

	//Get previous position
	glm::ivec2 previous_relative = message->_relative;

	//exit if at (range, range)
	if (previous_relative.x == (range))
        if (previous_relative.y == (range))
		    return nullptr;

	//calculate next message relative position
	glm::ivec2 next_relative = previous_relative;
	next_relative.x += 1;
	if ((next_relative.x)>range){
		next_relative.x = -range;
		next_relative.y = previous_relative.y + 1;
	}

	//skip own message
	if (next_relative.x == 0)
        if (next_relative.y == 0)
		    next_relative.x += 1;

	glm::ivec2 global_position;
	global_position.x =	sWRAP(message->_position.x + next_relative.x, width);
	global_position.y = sWRAP(message->_position.y + next_relative.y, width);

	int index = ((global_position.y)* width) + (global_position.x);
	
	xmachine_message_navmap_cell temp_message;
	temp_message._position = message->_position;
	temp_message._relative = next_relative;

	temp_message.x = tex1Dfetch(tex_xmachine_message_navmap_cell_x, index + d_tex_xmachine_message_navmap_cell_x_offset);	temp_message.y = tex1Dfetch(tex_xmachine_message_navmap_cell_y, index + d_tex_xmachine_message_navmap_cell_y_offset);	temp_message.exit_no = tex1Dfetch(tex_xmachine_message_navmap_cell_exit_no, index + d_tex_xmachine_message_navmap_cell_exit_no_offset);	temp_message.height = tex1Dfetch(tex_xmachine_message_navmap_cell_height, index + d_tex_xmachine_message_navmap_cell_height_offset);	temp_message.collision_x = tex1Dfetch(tex_xmachine_message_navmap_cell_collision_x, index + d_tex_xmachine_message_navmap_cell_collision_x_offset);	temp_message.collision_y = tex1Dfetch(tex_xmachine_message_navmap_cell_collision_y, index + d_tex_xmachine_message_navmap_cell_collision_y_offset);	

	message_share[threadIdx.x] = temp_message;

	return &message_share[threadIdx.x];
}

//method used by discrete agents accessing discrete messages to load messages into shared memory
__device__ void navmap_cell_message_to_sm(xmachine_message_navmap_cell_list* messages, char* message_share, int sm_index, int global_index){
		xmachine_message_navmap_cell temp_message;
		
		temp_message.x = messages->x[global_index];		
		temp_message.y = messages->y[global_index];		
		temp_message.exit_no = messages->exit_no[global_index];		
		temp_message.height = messages->height[global_index];		
		temp_message.collision_x = messages->collision_x[global_index];		
		temp_message.collision_y = messages->collision_y[global_index];		

	  int message_index = SHARE_INDEX(sm_index, sizeof(xmachine_message_navmap_cell));
	  xmachine_message_navmap_cell* sm_message = ((xmachine_message_navmap_cell*)&message_share[message_index]);
	  sm_message[0] = temp_message;
}

//Get first navmap_cell message 
//Used by discrete agents this accesses messages with texture cache. Agent position is determined by position in the grid/block
//Possibility of upto 8 thread divergences
__device__ xmachine_message_navmap_cell* get_first_navmap_cell_message_discrete(xmachine_message_navmap_cell_list* messages){

	//shared memory get from offset dependant on sm usage in function
	extern __shared__ int sm_data [];

	char* message_share = (char*)&sm_data[0];
  
	__syncthreads();

	int range = d_message_navmap_cell_range;
	int width = d_message_navmap_cell_width;
	int sm_grid_width = blockDim.x + (range* 2);
	
	
	glm::ivec2 global_position;
	global_position.x = (blockIdx.x * blockDim.x) + threadIdx.x;
	global_position.y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = global_position.x + (global_position.y * width);
	

	//calculate the position in shared memory of first load
	glm::ivec2 sm_pos;
	sm_pos.x = threadIdx.x + range;
	sm_pos.y = threadIdx.y + range;
	int sm_index = (sm_pos.y * sm_grid_width) + sm_pos.x;

	//each thread loads to shared memory (coalesced read)
	navmap_cell_message_to_sm(messages, message_share, sm_index, index);

	//check for edge conditions
	int left_border = (threadIdx.x < range);
	int right_border = (threadIdx.x >= (blockDim.x-range));
	int top_border = (threadIdx.y < range);
	int bottom_border = (threadIdx.y >= (blockDim.y-range));

	
	int  border_index;
	int  sm_border_index;

	//left
	if (left_border){	
		glm::ivec2 border_index_2d = global_position;
		border_index_2d.x = sWRAP(border_index_2d.x - range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = (sm_pos.y * sm_grid_width) + threadIdx.x;
		
		navmap_cell_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	//right
	if (right_border){
		glm::ivec2 border_index_2d = global_position;
		border_index_2d.x = sWRAP(border_index_2d.x + range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = (sm_pos.y * sm_grid_width) + (sm_pos.x + range);

		navmap_cell_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	//top
	if (top_border){
		glm::ivec2 border_index_2d = global_position;
		border_index_2d.y = sWRAP(border_index_2d.y - range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = (threadIdx.y * sm_grid_width) + sm_pos.x;

		navmap_cell_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	//bottom
	if (bottom_border){
		glm::ivec2 border_index_2d = global_position;
		border_index_2d.y = sWRAP(border_index_2d.y + range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = ((sm_pos.y + range) * sm_grid_width) + sm_pos.x;

		navmap_cell_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	//top left
	if ((top_border)&&(left_border)){	
		glm::ivec2 border_index_2d = global_position;
		border_index_2d.x = sWRAP(border_index_2d.x - range, width);
		border_index_2d.y = sWRAP(border_index_2d.y - range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = (threadIdx.y * sm_grid_width) + threadIdx.x;
		
		navmap_cell_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	//top right
	if ((top_border)&&(right_border)){	
		glm::ivec2 border_index_2d = global_position;
		border_index_2d.x = sWRAP(border_index_2d.x + range, width);
		border_index_2d.y = sWRAP(border_index_2d.y - range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = (threadIdx.y * sm_grid_width) + (sm_pos.x + range);
		
		navmap_cell_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	//bottom right
	if ((bottom_border)&&(right_border)){	
		glm::ivec2 border_index_2d = global_position;
		border_index_2d.x = sWRAP(border_index_2d.x + range, width);
		border_index_2d.y = sWRAP(border_index_2d.y + range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = ((sm_pos.y + range) * sm_grid_width) + (sm_pos.x + range);
		
		navmap_cell_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	//bottom left
	if ((bottom_border)&&(left_border)){	
		glm::ivec2 border_index_2d = global_position;
		border_index_2d.x = sWRAP(border_index_2d.x - range, width);
		border_index_2d.y = sWRAP(border_index_2d.y + range, width);
		border_index = (border_index_2d.y * width) + border_index_2d.x;
		sm_border_index = ((sm_pos.y + range) * sm_grid_width) + threadIdx.x;
		
		navmap_cell_message_to_sm(messages, message_share, sm_border_index, border_index);
	}

	__syncthreads();
	
  
	//top left of block position sm index
	sm_index = (threadIdx.y * sm_grid_width) + threadIdx.x;
	
	int message_index = SHARE_INDEX(sm_index, sizeof(xmachine_message_navmap_cell));
	xmachine_message_navmap_cell* temp = ((xmachine_message_navmap_cell*)&message_share[message_index]);
	temp->_relative = glm::ivec2(-range, -range); //this is the relative position
	return temp;
}

//Get next navmap_cell message 
//Used by discrete agents this accesses messages through shared memory which were all loaded on first message retrieval call.
__device__ xmachine_message_navmap_cell* get_next_navmap_cell_message_discrete(xmachine_message_navmap_cell* message, xmachine_message_navmap_cell_list* messages){

	//shared memory get from offset dependant on sm usage in function
	extern __shared__ int sm_data [];

	char* message_share = (char*)&sm_data[0];
  
	__syncthreads();
	
	int range = d_message_navmap_cell_range;
	int sm_grid_width = blockDim.x+(range*2);


	//Get previous position
	glm::ivec2 previous_relative = message->_relative;

	//exit if at (range, range)
	if (previous_relative.x == range)
        if (previous_relative.y == range)
		    return nullptr;

	//calculate next message relative position
	glm::ivec2 next_relative = previous_relative;
	next_relative.x += 1;
	if ((next_relative.x)>range){
		next_relative.x = -range;
		next_relative.y = previous_relative.y + 1;
	}

	//skip own message
	if (next_relative.x == 0)
        if (next_relative.y == 0)
		    next_relative.x += 1;


	//calculate the next message position
	glm::ivec2 next_position;// = block_position+next_relative;
	//offset next position by the sm border size
	next_position.x = threadIdx.x + next_relative.x + range;
	next_position.y = threadIdx.y + next_relative.y + range;

	int sm_index = next_position.x + (next_position.y * sm_grid_width);
	
	__syncthreads();
  
	int message_index = SHARE_INDEX(sm_index, sizeof(xmachine_message_navmap_cell));
	xmachine_message_navmap_cell* temp = ((xmachine_message_navmap_cell*)&message_share[message_index]);
	temp->_relative = next_relative; //this is the relative position
	return temp;
}

//Get first navmap_cell message
template <int AGENT_TYPE>
__device__ xmachine_message_navmap_cell* get_first_navmap_cell_message(xmachine_message_navmap_cell_list* messages, int agent_x, int agent_y){

	if (AGENT_TYPE == DISCRETE_2D)	//use shared memory method
		return get_first_navmap_cell_message_discrete(messages);
	else	//use texture fetching method
		return get_first_navmap_cell_message_continuous(messages, agent_x, agent_y);

}

//Get next navmap_cell message
template <int AGENT_TYPE>
__device__ xmachine_message_navmap_cell* get_next_navmap_cell_message(xmachine_message_navmap_cell* message, xmachine_message_navmap_cell_list* messages){

	if (AGENT_TYPE == DISCRETE_2D)	//use shared memory method
		return get_next_navmap_cell_message_discrete(message, messages);
	else	//use texture fetching method
		return get_next_navmap_cell_message_continuous(message, messages);

}

	
/////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created GPU kernels  */



/**
 *
 */
__global__ void GPUFLAME_output_pedestrian_location(xmachine_memory_agent_list* agents, xmachine_message_pedestrian_location_list* pedestrian_location_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_agent_count)
        return;
    

	//SoA to AoS - xmachine_memory_output_pedestrian_location Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_agent agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.velx = agents->velx[index];
	agent.vely = agents->vely[index];
	agent.steer_x = agents->steer_x[index];
	agent.steer_y = agents->steer_y[index];
	agent.height = agents->height[index];
	agent.exit_no = agents->exit_no[index];
	agent.speed = agents->speed[index];
	agent.lod = agents->lod[index];
	agent.animate = agents->animate[index];
	agent.animate_dir = agents->animate_dir[index];
	agent.estado = agents->estado[index];
	agent.tick = agents->tick[index];
	agent.estado_movimiento = agents->estado_movimiento[index];

	//FLAME function call
	int dead = !output_pedestrian_location(&agent, pedestrian_location_messages	);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_output_pedestrian_location Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->velx[index] = agent.velx;
	agents->vely[index] = agent.vely;
	agents->steer_x[index] = agent.steer_x;
	agents->steer_y[index] = agent.steer_y;
	agents->height[index] = agent.height;
	agents->exit_no[index] = agent.exit_no;
	agents->speed[index] = agent.speed;
	agents->lod[index] = agent.lod;
	agents->animate[index] = agent.animate;
	agents->animate_dir[index] = agent.animate_dir;
	agents->estado[index] = agent.estado;
	agents->tick[index] = agent.tick;
	agents->estado_movimiento[index] = agent.estado_movimiento;
}

/**
 *
 */
__global__ void GPUFLAME_avoid_pedestrians(xmachine_memory_agent_list* agents, xmachine_message_pedestrian_location_list* pedestrian_location_messages, xmachine_message_pedestrian_location_PBM* partition_matrix, RNG_rand48* rand48){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_agent_count)
        return;
    

	//SoA to AoS - xmachine_memory_avoid_pedestrians Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_agent agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.velx = agents->velx[index];
	agent.vely = agents->vely[index];
	agent.steer_x = agents->steer_x[index];
	agent.steer_y = agents->steer_y[index];
	agent.height = agents->height[index];
	agent.exit_no = agents->exit_no[index];
	agent.speed = agents->speed[index];
	agent.lod = agents->lod[index];
	agent.animate = agents->animate[index];
	agent.animate_dir = agents->animate_dir[index];
	agent.estado = agents->estado[index];
	agent.tick = agents->tick[index];
	agent.estado_movimiento = agents->estado_movimiento[index];

	//FLAME function call
	int dead = !avoid_pedestrians(&agent, pedestrian_location_messages, partition_matrix, rand48);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_avoid_pedestrians Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->velx[index] = agent.velx;
	agents->vely[index] = agent.vely;
	agents->steer_x[index] = agent.steer_x;
	agents->steer_y[index] = agent.steer_y;
	agents->height[index] = agent.height;
	agents->exit_no[index] = agent.exit_no;
	agents->speed[index] = agent.speed;
	agents->lod[index] = agent.lod;
	agents->animate[index] = agent.animate;
	agents->animate_dir[index] = agent.animate_dir;
	agents->estado[index] = agent.estado;
	agents->tick[index] = agent.tick;
	agents->estado_movimiento[index] = agent.estado_movimiento;
}

/**
 *
 */
__global__ void GPUFLAME_output_pedestrian_state(xmachine_memory_agent_list* agents, xmachine_message_pedestrian_state_list* pedestrian_state_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_agent_count)
        return;
    

	//SoA to AoS - xmachine_memory_output_pedestrian_state Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_agent agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.velx = agents->velx[index];
	agent.vely = agents->vely[index];
	agent.steer_x = agents->steer_x[index];
	agent.steer_y = agents->steer_y[index];
	agent.height = agents->height[index];
	agent.exit_no = agents->exit_no[index];
	agent.speed = agents->speed[index];
	agent.lod = agents->lod[index];
	agent.animate = agents->animate[index];
	agent.animate_dir = agents->animate_dir[index];
	agent.estado = agents->estado[index];
	agent.tick = agents->tick[index];
	agent.estado_movimiento = agents->estado_movimiento[index];

	//FLAME function call
	int dead = !output_pedestrian_state(&agent, pedestrian_state_messages	);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_output_pedestrian_state Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->velx[index] = agent.velx;
	agents->vely[index] = agent.vely;
	agents->steer_x[index] = agent.steer_x;
	agents->steer_y[index] = agent.steer_y;
	agents->height[index] = agent.height;
	agents->exit_no[index] = agent.exit_no;
	agents->speed[index] = agent.speed;
	agents->lod[index] = agent.lod;
	agents->animate[index] = agent.animate;
	agents->animate_dir[index] = agent.animate_dir;
	agents->estado[index] = agent.estado;
	agents->tick[index] = agent.tick;
	agents->estado_movimiento[index] = agent.estado_movimiento;
}

/**
 *
 */
__global__ void GPUFLAME_infect_pedestrians(xmachine_memory_agent_list* agents, xmachine_message_pedestrian_state_list* pedestrian_state_messages, xmachine_message_pedestrian_state_PBM* partition_matrix, RNG_rand48* rand48){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_agent_count)
        return;
    

	//SoA to AoS - xmachine_memory_infect_pedestrians Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_agent agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.velx = agents->velx[index];
	agent.vely = agents->vely[index];
	agent.steer_x = agents->steer_x[index];
	agent.steer_y = agents->steer_y[index];
	agent.height = agents->height[index];
	agent.exit_no = agents->exit_no[index];
	agent.speed = agents->speed[index];
	agent.lod = agents->lod[index];
	agent.animate = agents->animate[index];
	agent.animate_dir = agents->animate_dir[index];
	agent.estado = agents->estado[index];
	agent.tick = agents->tick[index];
	agent.estado_movimiento = agents->estado_movimiento[index];

	//FLAME function call
	int dead = !infect_pedestrians(&agent, pedestrian_state_messages, partition_matrix, rand48);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_infect_pedestrians Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->velx[index] = agent.velx;
	agents->vely[index] = agent.vely;
	agents->steer_x[index] = agent.steer_x;
	agents->steer_y[index] = agent.steer_y;
	agents->height[index] = agent.height;
	agents->exit_no[index] = agent.exit_no;
	agents->speed[index] = agent.speed;
	agents->lod[index] = agent.lod;
	agents->animate[index] = agent.animate;
	agents->animate_dir[index] = agent.animate_dir;
	agents->estado[index] = agent.estado;
	agents->tick[index] = agent.tick;
	agents->estado_movimiento[index] = agent.estado_movimiento;
}

/**
 *
 */
__global__ void GPUFLAME_move(xmachine_memory_agent_list* agents, xmachine_message_check_in_list* check_in_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_agent_count)
        return;
    

	//SoA to AoS - xmachine_memory_move Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_agent agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.velx = agents->velx[index];
	agent.vely = agents->vely[index];
	agent.steer_x = agents->steer_x[index];
	agent.steer_y = agents->steer_y[index];
	agent.height = agents->height[index];
	agent.exit_no = agents->exit_no[index];
	agent.speed = agents->speed[index];
	agent.lod = agents->lod[index];
	agent.animate = agents->animate[index];
	agent.animate_dir = agents->animate_dir[index];
	agent.estado = agents->estado[index];
	agent.tick = agents->tick[index];
	agent.estado_movimiento = agents->estado_movimiento[index];

	//FLAME function call
	int dead = !move(&agent, check_in_messages	);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_move Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->velx[index] = agent.velx;
	agents->vely[index] = agent.vely;
	agents->steer_x[index] = agent.steer_x;
	agents->steer_y[index] = agent.steer_y;
	agents->height[index] = agent.height;
	agents->exit_no[index] = agent.exit_no;
	agents->speed[index] = agent.speed;
	agents->lod[index] = agent.lod;
	agents->animate[index] = agent.animate;
	agents->animate_dir[index] = agent.animate_dir;
	agents->estado[index] = agent.estado;
	agents->tick[index] = agent.tick;
	agents->estado_movimiento[index] = agent.estado_movimiento;
}

/**
 *
 */
__global__ void GPUFLAME_prueba(xmachine_memory_medic_list* agents){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_medic_count)
        return;
    

	//SoA to AoS - xmachine_memory_prueba Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_medic agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.x = agents->x[index];
	agent.y = agents->y[index];

	//FLAME function call
	int dead = !prueba(&agent);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_prueba Coalesced memory write (ignore arrays)
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
}

/**
 *
 */
__global__ void GPUFLAME_receptionServer(xmachine_memory_receptionist_list* agents, xmachine_message_check_in_list* check_in_messages, xmachine_message_avisar_paciente_list* avisar_paciente_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    
    //No partitioned input requires threads to be launched beyond the agent count to ensure full block sizes
    

	//SoA to AoS - xmachine_memory_receptionServer Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_receptionist agent;
    //No partitioned input may launch more threads than required - only load agent data within bounds. 
    if (index < d_xmachine_memory_receptionist_count){
    
	agent.x = agents->x[index];
	agent.y = agents->y[index];
    agent.colaPacientes = &(agents->colaPacientes[index]);
	agent.front = agents->front[index];
	agent.rear = agents->rear[index];
	agent.size = agents->size[index];
	} else {
	
	agent.x = 0;
	agent.y = 0;
    agent.colaPacientes = nullptr;
	agent.front = 0;
	agent.rear = 0;
	agent.size = 0;
	}

	//FLAME function call
	int dead = !receptionServer(&agent, check_in_messages, avisar_paciente_messages	);
	

	
    //No partitioned input may launch more threads than required - only write agent data within bounds. 
    if (index < d_xmachine_memory_receptionist_count){
    //continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_receptionServer Coalesced memory write (ignore arrays)
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->front[index] = agent.front;
	agents->rear[index] = agent.rear;
	agents->size[index] = agent.size;
	}
}

/**
 *
 */
__global__ void GPUFLAME_output_navmap_cells(xmachine_memory_navmap_list* agents, xmachine_message_navmap_cell_list* navmap_cell_messages){
	
	
	//discrete agent: index is position in 2D agent grid
	int width = (blockDim.x * gridDim.x);
	glm::ivec2 global_position;
	global_position.x = (blockIdx.x * blockDim.x) + threadIdx.x;
	global_position.y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = global_position.x + (global_position.y * width);
	

	//SoA to AoS - xmachine_memory_output_navmap_cells Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_navmap agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.exit_no = agents->exit_no[index];
	agent.height = agents->height[index];
	agent.collision_x = agents->collision_x[index];
	agent.collision_y = agents->collision_y[index];
	agent.exit0_x = agents->exit0_x[index];
	agent.exit0_y = agents->exit0_y[index];
	agent.exit1_x = agents->exit1_x[index];
	agent.exit1_y = agents->exit1_y[index];
	agent.exit2_x = agents->exit2_x[index];
	agent.exit2_y = agents->exit2_y[index];
	agent.exit3_x = agents->exit3_x[index];
	agent.exit3_y = agents->exit3_y[index];
	agent.exit4_x = agents->exit4_x[index];
	agent.exit4_y = agents->exit4_y[index];
	agent.exit5_x = agents->exit5_x[index];
	agent.exit5_y = agents->exit5_y[index];
	agent.exit6_x = agents->exit6_x[index];
	agent.exit6_y = agents->exit6_y[index];
	agent.cant_generados = agents->cant_generados[index];

	//FLAME function call
	output_navmap_cells(&agent, navmap_cell_messages	);
	

	

	//AoS to SoA - xmachine_memory_output_navmap_cells Coalesced memory write (ignore arrays)
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->exit_no[index] = agent.exit_no;
	agents->height[index] = agent.height;
	agents->collision_x[index] = agent.collision_x;
	agents->collision_y[index] = agent.collision_y;
	agents->exit0_x[index] = agent.exit0_x;
	agents->exit0_y[index] = agent.exit0_y;
	agents->exit1_x[index] = agent.exit1_x;
	agents->exit1_y[index] = agent.exit1_y;
	agents->exit2_x[index] = agent.exit2_x;
	agents->exit2_y[index] = agent.exit2_y;
	agents->exit3_x[index] = agent.exit3_x;
	agents->exit3_y[index] = agent.exit3_y;
	agents->exit4_x[index] = agent.exit4_x;
	agents->exit4_y[index] = agent.exit4_y;
	agents->exit5_x[index] = agent.exit5_x;
	agents->exit5_y[index] = agent.exit5_y;
	agents->exit6_x[index] = agent.exit6_x;
	agents->exit6_y[index] = agent.exit6_y;
	agents->cant_generados[index] = agent.cant_generados;
}

/**
 *
 */
__global__ void GPUFLAME_generate_pedestrians(xmachine_memory_navmap_list* agents, xmachine_memory_agent_list* agent_agents, RNG_rand48* rand48){
	
	
	//discrete agent: index is position in 2D agent grid
	int width = (blockDim.x * gridDim.x);
	glm::ivec2 global_position;
	global_position.x = (blockIdx.x * blockDim.x) + threadIdx.x;
	global_position.y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = global_position.x + (global_position.y * width);
	

	//SoA to AoS - xmachine_memory_generate_pedestrians Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_navmap agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.exit_no = agents->exit_no[index];
	agent.height = agents->height[index];
	agent.collision_x = agents->collision_x[index];
	agent.collision_y = agents->collision_y[index];
	agent.exit0_x = agents->exit0_x[index];
	agent.exit0_y = agents->exit0_y[index];
	agent.exit1_x = agents->exit1_x[index];
	agent.exit1_y = agents->exit1_y[index];
	agent.exit2_x = agents->exit2_x[index];
	agent.exit2_y = agents->exit2_y[index];
	agent.exit3_x = agents->exit3_x[index];
	agent.exit3_y = agents->exit3_y[index];
	agent.exit4_x = agents->exit4_x[index];
	agent.exit4_y = agents->exit4_y[index];
	agent.exit5_x = agents->exit5_x[index];
	agent.exit5_y = agents->exit5_y[index];
	agent.exit6_x = agents->exit6_x[index];
	agent.exit6_y = agents->exit6_y[index];
	agent.cant_generados = agents->cant_generados[index];

	//FLAME function call
	generate_pedestrians(&agent, agent_agents, rand48);
	

	

	//AoS to SoA - xmachine_memory_generate_pedestrians Coalesced memory write (ignore arrays)
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->exit_no[index] = agent.exit_no;
	agents->height[index] = agent.height;
	agents->collision_x[index] = agent.collision_x;
	agents->collision_y[index] = agent.collision_y;
	agents->exit0_x[index] = agent.exit0_x;
	agents->exit0_y[index] = agent.exit0_y;
	agents->exit1_x[index] = agent.exit1_x;
	agents->exit1_y[index] = agent.exit1_y;
	agents->exit2_x[index] = agent.exit2_x;
	agents->exit2_y[index] = agent.exit2_y;
	agents->exit3_x[index] = agent.exit3_x;
	agents->exit3_y[index] = agent.exit3_y;
	agents->exit4_x[index] = agent.exit4_x;
	agents->exit4_y[index] = agent.exit4_y;
	agents->exit5_x[index] = agent.exit5_x;
	agents->exit5_y[index] = agent.exit5_y;
	agents->exit6_x[index] = agent.exit6_x;
	agents->exit6_y[index] = agent.exit6_y;
	agents->cant_generados[index] = agent.cant_generados;
}

/**
 *
 */
__global__ void GPUFLAME_generate_medics(xmachine_memory_navmap_list* agents, xmachine_memory_medic_list* medic_agents, RNG_rand48* rand48){
	
	
	//discrete agent: index is position in 2D agent grid
	int width = (blockDim.x * gridDim.x);
	glm::ivec2 global_position;
	global_position.x = (blockIdx.x * blockDim.x) + threadIdx.x;
	global_position.y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = global_position.x + (global_position.y * width);
	

	//SoA to AoS - xmachine_memory_generate_medics Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_navmap agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.x = agents->x[index];
	agent.y = agents->y[index];
	agent.exit_no = agents->exit_no[index];
	agent.height = agents->height[index];
	agent.collision_x = agents->collision_x[index];
	agent.collision_y = agents->collision_y[index];
	agent.exit0_x = agents->exit0_x[index];
	agent.exit0_y = agents->exit0_y[index];
	agent.exit1_x = agents->exit1_x[index];
	agent.exit1_y = agents->exit1_y[index];
	agent.exit2_x = agents->exit2_x[index];
	agent.exit2_y = agents->exit2_y[index];
	agent.exit3_x = agents->exit3_x[index];
	agent.exit3_y = agents->exit3_y[index];
	agent.exit4_x = agents->exit4_x[index];
	agent.exit4_y = agents->exit4_y[index];
	agent.exit5_x = agents->exit5_x[index];
	agent.exit5_y = agents->exit5_y[index];
	agent.exit6_x = agents->exit6_x[index];
	agent.exit6_y = agents->exit6_y[index];
	agent.cant_generados = agents->cant_generados[index];

	//FLAME function call
	generate_medics(&agent, medic_agents, rand48);
	

	

	//AoS to SoA - xmachine_memory_generate_medics Coalesced memory write (ignore arrays)
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
	agents->exit_no[index] = agent.exit_no;
	agents->height[index] = agent.height;
	agents->collision_x[index] = agent.collision_x;
	agents->collision_y[index] = agent.collision_y;
	agents->exit0_x[index] = agent.exit0_x;
	agents->exit0_y[index] = agent.exit0_y;
	agents->exit1_x[index] = agent.exit1_x;
	agents->exit1_y[index] = agent.exit1_y;
	agents->exit2_x[index] = agent.exit2_x;
	agents->exit2_y[index] = agent.exit2_y;
	agents->exit3_x[index] = agent.exit3_x;
	agents->exit3_y[index] = agent.exit3_y;
	agents->exit4_x[index] = agent.exit4_x;
	agents->exit4_y[index] = agent.exit4_y;
	agents->exit5_x[index] = agent.exit5_x;
	agents->exit5_y[index] = agent.exit5_y;
	agents->exit6_x[index] = agent.exit6_x;
	agents->exit6_y[index] = agent.exit6_y;
	agents->cant_generados[index] = agent.cant_generados;
}

	
	
/* Graph utility functions */



/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Rand48 functions */

__device__ static glm::uvec2 RNG_rand48_iterate_single(glm::uvec2 Xn, glm::uvec2 A, glm::uvec2 C)
{
	unsigned int R0, R1;

	// low 24-bit multiplication
	const unsigned int lo00 = __umul24(Xn.x, A.x);
	const unsigned int hi00 = __umulhi(Xn.x, A.x);

	// 24bit distribution of 32bit multiplication results
	R0 = (lo00 & 0xFFFFFF);
	R1 = (lo00 >> 24) | (hi00 << 8);

	R0 += C.x; R1 += C.y;

	// transfer overflows
	R1 += (R0 >> 24);
	R0 &= 0xFFFFFF;

	// cross-terms, low/hi 24-bit multiplication
	R1 += __umul24(Xn.y, A.x);
	R1 += __umul24(Xn.x, A.y);

	R1 &= 0xFFFFFF;

	return glm::uvec2(R0, R1);
}

//Templated function
template <int AGENT_TYPE>
__device__ float rnd(RNG_rand48* rand48){

	int index;
	
	//calculate the agents index in global agent list
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x * gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x * blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y * blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y * width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	glm::uvec2 state = rand48->seeds[index];
	glm::uvec2 A = rand48->A;
	glm::uvec2 C = rand48->C;

	int rand = ( state.x >> 17 ) | ( state.y << 7);

	// this actually iterates the RNG
	state = RNG_rand48_iterate_single(state, A, C);

	rand48->seeds[index] = state;

	return (float)rand/2147483647;
}

__device__ float rnd(RNG_rand48* rand48){
	return rnd<DISCRETE_2D>(rand48);
}

#endif //_FLAMEGPU_KERNELS_H_
